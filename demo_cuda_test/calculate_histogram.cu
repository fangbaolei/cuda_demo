#include "hip/hip_runtime.h"
#include "funset.hpp"
#include <iostream>
#include <algorithm>
#include <memory>
#include <vector>
#include <hip/hip_runtime.h> // For the CUDA runtime routines (prefixed with "cuda_")
#include <>
#include "common.hpp"

/* __global__: ���������޶���;���豸������;�������˵���,��������3.2�����Ͽ�����
�豸�˵���;�����ĺ����ķ���ֵ������void����;�Դ����ͺ����ĵ������첽��,����
�豸��ȫ�����������֮ǰ�ͷ�����;�Դ����ͺ����ĵ��ñ���ָ��ִ������,��������
�豸��ִ�к���ʱ��grid��block��ά��,�Լ���ص���(������<<<   >>>�����);
a kernel,��ʾ�˺���Ϊ�ں˺���(������GPU�ϵ�CUDA���м��㺯����Ϊkernel(�ں˺�
��),�ں˺�������ͨ��__global__���������޶�������); */
__global__ static void calculate_histogram(const unsigned char* data, int length, unsigned int* hist)
{
	/* __shared__: ���������޶�����ʹ��__shared__�޶�����������__device__��
	�������ã���ʱ�����ı���λ��block�еĹ���洢���ռ��У���block������ͬ
	���������ڣ�����ͨ��block�ڵ������̷߳��ʣ�__shared__��__constant__����
	Ĭ��Ϊ�Ǿ�̬�洢����__shared__ǰ���Լ�extern�ؼ��֣�����ʾ���Ǳ�����С
	��ִ�в���ȷ����__shared__����������ʱ���ܳ�ʼ�������Խ�CUDA C�Ĺؼ���
	__shared__��ӵ����������У��⽫ʹ�������פ���ڹ����ڴ��У�CUDA C����
	���Թ����ڴ��еı�������ͨ�������ֱ��ȡ��ͬ�Ĵ���ʽ */
	// clear out the accumulation buffer called temp since we are launched with
	// 256 threads, it is easy to clear that memory with one write per thread
	__shared__  unsigned int temp[256]; // �����ڴ滺����
	temp[threadIdx.x] = 0;
	/* __syncthreads: ���߳̿��е��߳̽���ͬ����CUDA�ܹ���ȷ���������߳̿�
	�е�ÿ���̶߳�ִ����__syncthreads()������û���κ��߳���ִ��
	__syncthreads()֮���ָ��;��ͬһ��block�е��߳�ͨ������洢��(shared
	memory)�������ݣ���ͨ��դ��ͬ��(������kernel��������Ҫͬ����λ�õ���
	__syncthreads()����)��֤�̼߳��ܹ���ȷ�ع������ݣ�ʹ��clock()������ʱ��
	���ں˺�����Ҫ������һ�δ���Ŀ�ʼ�ͽ�����λ�÷ֱ����һ��clock()������
	���������¼���������ڵ���__syncthreads()������һ��block�е�����
	thread��Ҫ��ʱ������ͬ�ģ����ֻ��Ҫ��¼ÿ��blockִ����Ҫ��ʱ������ˣ�
	������Ҫ��¼ÿ��thread��ʱ�� */
	__syncthreads();

	/* gridDim: ���ñ���,���������߳������ά��,���������߳̿���˵,���
	������һ������,���������̸߳�ÿһά�Ĵ�С,��ÿ���̸߳����߳̿������.
	Ϊdim3���ͣ�
	blockDim: ���ñ���,����˵��ÿ��block��ά����ߴ�.Ϊdim3����,����
	��block������ά���ϵĳߴ���Ϣ;���������߳̿���˵,���������һ������,
	��������߳̿���ÿһά���߳�����;
	blockIdx: ���ñ���,�����а�����ֵ���ǵ�ǰִ���豸������߳̿������;��
	��˵����ǰthread���ڵ�block������grid�е�λ��,blockIdx.xȡֵ��Χ��
	[0,gridDim.x-1],blockIdx.yȡֵ��Χ��[0, gridDim.y-1].Ϊuint3����,
	������һ��block��grid�и���ά���ϵ�������Ϣ;
	threadIdx: ���ñ���,�����а�����ֵ���ǵ�ǰִ���豸������߳�����;����
	˵����ǰthread��block�е�λ��;����߳���һά�Ŀɻ�ȡthreadIdx.x,���
	�Ƕ�ά�Ļ��ɻ�ȡthreadIdx.y,�������ά�Ļ��ɻ�ȡthreadIdx.z;Ϊuint3��
	��,������һ��thread��block�и���ά�ȵ�������Ϣ */
	// calculate the starting index and the offset to the next block that each thread will be processing
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while (i < length) {
		/* atomicAdd: ԭ�Ӳ���,�ײ�Ӳ����ȷ����ִ����Щԭ�Ӳ���ʱ����
		���κ��̶߳������ȡ��д���ַaddr�ϵ�ֵ��ԭ�Ӻ���(atomic
		function)��λ��ȫ�ֻ���洢����һ��32λ��64λ��ִ��
		read-modify-write��ԭ�Ӳ�����Ҳ����˵��������߳�ͬʱ����ȫ�ֻ�
		����洢����ͬһλ��ʱ����֤ÿ���߳��ܹ�ʵ�ֶԹ����д���ݵĻ�
		���������һ���������֮ǰ�������κ��̶߳��޷����ʴ˵�ַ��֮��
		�Խ���һ���̳�Ϊԭ�Ӳ���������Ϊÿ���̵߳Ĳ���������Ӱ�쵽����
		�̡߳����仰˵��ԭ�Ӳ����ܹ���֤��һ����ַ�ĵ�ǰ�������֮ǰ��
		�����̶߳����ܷ��������ַ��
		atomicAdd(addr,y)��������һ��ԭ�ӵĲ������У�����������а�����
		ȡ��ַaddr����ֵ����y���ӵ����ֵ���Լ����������ص�ַaddr�� */
		atomicAdd(&temp[data[i]], 1);
		i += stride;
	}

	// sync the data from the above writes to shared memory then add the shared memory values to the values from
	// the other thread blocks using global memory atomic adds same as before, since we have 256 threads,
	// updating the global histogram is just one write per thread!
	__syncthreads();
	// ��ÿ���߳̿��ֱ��ͼ�ϲ�Ϊ�������յ�ֱ��ͼ
	atomicAdd(&(hist[threadIdx.x]), temp[threadIdx.x]);
}

int calculate_histogram_gpu(const unsigned char* data, int length, unsigned int* hist, unsigned int& value, float* elapsed_time)
{
	/* hipEvent_t: CUDA event types,�ṹ������, CUDA�¼�,���ڲ���GPU��ĳ
	�������ϻ��ѵ�ʱ��,CUDA�е��¼���������һ��GPUʱ���,����CUDA�¼�����
	GPU��ʵ�ֵ�,������ǲ����ڶ�ͬʱ�����豸�������������Ļ�ϴ����ʱ */
	hipEvent_t start, stop;
	// hipEventCreate: ����һ���¼�����,�첽����
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// hipEventRecord: ��¼һ���¼�,�첽����,start��¼��ʼʱ��
	hipEventRecord(start, 0);

	unsigned char* dev_buffer{ nullptr };
	unsigned int* dev_hist{ nullptr };

	// hipMalloc: ���豸�˷����ڴ�
	hipMalloc(&dev_buffer, length);
	hipMalloc(&dev_hist, 256 * sizeof(unsigned int));

	/* hipMemcpy: �������˺��豸�˿�������,�˺������ĸ���������������֮һ:
	(1). hipMemcpyHostToHost: �������ݴ������˵�������
	(2). hipMemcpyHostToDevice: �������ݴ������˵��豸��
	(3). hipMemcpyDeviceToHost: �������ݴ��豸�˵�������
	(4). hipMemcpyDeviceToDevice: �������ݴ��豸�˵��豸��
	(5). hipMemcpyDefault: ��ָ��ֵ�Զ��ƶϿ������ݷ���,��Ҫ֧��
	ͳһ����Ѱַ(CUDA6.0�����ϰ汾)
	hipMemcpy��������������ͬ���� */
	hipMemcpy(dev_buffer, data, length, hipMemcpyHostToDevice);

	/* hipMemset: �洢����ʼ������,��GPU�ڴ���ִ�С���ָ����ֵ��ʼ��������
	�豸�ڴ� */
	hipMemset(dev_hist, 0, 256 * sizeof(unsigned int));

	// hipDeviceProp_t: cuda�豸���Խṹ��
	// kernel launch - 2x the number of mps gave best timing
	hipDeviceProp_t prop;
	// hipGetDeviceProperties: ��ȡGPU�豸�����Ϣ
	hipGetDeviceProperties(&prop, 0);
	// hipDeviceProp_t::multiProcessorCount: �豸�϶ദ����������
	int blocks = prop.multiProcessorCount;
	fprintf(stderr, "multiProcessorCount: %d\n", blocks);

	/* <<< >>>: ΪCUDA����������,ָ���߳�������߳̿�ά�ȵ�,����ִ�в�
	����CUDA������������ʱϵͳ,����˵���ں˺����е��߳�����,�Լ��߳������
	��֯��;����������Щ���������Ǵ��ݸ��豸����Ĳ���,���Ǹ�������ʱ���
	�����豸����,���ݸ��豸���뱾��Ĳ����Ƿ���Բ�����д��ݵ�,�����׼�ĺ�
	������һ��;��ͬ�����������豸���̵߳���������֯��ʽ�в�ͬ��Լ��;����
	��Ϊkernel���õ�����������������㹻�Ŀռ�,�ٵ���kernel����,������
	GPU����ʱ�ᷢ������,����Խ���;
	ʹ������ʱAPIʱ,��Ҫ�ڵ��õ��ں˺�����������б�ֱ����<<<Dg,Db,Ns,S>>>
	����ʽ����ִ������,���У�Dg��һ��dim3�ͱ���,��������grid��ά�Ⱥ͸���
	ά���ϵĳߴ�.���ú�Dg��,grid�н���Dg.x*Dg.y*Dg.z��block;Db��
	һ��dim3�ͱ���,��������block��ά�Ⱥ͸���ά���ϵĳߴ�.���ú�Db��,ÿ��
	block�н���Db.x*Db.y*Db.z��thread;Ns��һ��unsigned int�ͱ���,ָ������Ϊ�˵�
	�ö�̬����Ĺ���洢����С,��Щ��̬����Ĵ洢���ɹ�����Ϊ�ⲿ����
	(extern __shared__)�������κα���ʹ��;Ns��һ����ѡ����,Ĭ��ֵΪ0;SΪ
	hipStream_t����,�����������ں˺�����������.S��һ����ѡ����,Ĭ��ֵ0. */
	// ���߳̿������ΪGPU�д�����������2��ʱ�����ﵽ��������
	calculate_histogram << <blocks * 2, 256 >> >(dev_buffer, length, dev_hist);

	hipMemcpy(hist, dev_hist, 256 * sizeof(unsigned int), hipMemcpyDeviceToHost);

	value = 0;
	for (int i = 0; i < 256; ++i) {
		value += hist[i];
	}

	// hipFree: �ͷ��豸����hipMalloc����������ڴ�
	hipFree(dev_buffer);
	hipFree(dev_hist);

	// hipEventRecord: ��¼һ���¼�,�첽����,stop��¼����ʱ��
	hipEventRecord(stop, 0);
	// hipEventSynchronize: �¼�ͬ��,�ȴ�һ���¼����,�첽����
	hipEventSynchronize(stop);
	// cudaEventElapseTime: ���������¼�֮�侭����ʱ��,��λΪ����,�첽����
	hipEventElapsedTime(elapsed_time, start, stop);
	// hipEventDestroy: �����¼�����,�첽����
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}
