#include "hip/hip_runtime.h"
#include "funset.hpp"
#include <iostream>
#include <algorithm>
#include <memory>
#include <hip/hip_runtime.h> // For the CUDA runtime routines (prefixed with "cuda_")
#include <>
#include "common.hpp"

/* __global__: ���������޶���;���豸������;�������˵���,��������3.2�����Ͽ�����
�豸�˵���;�����ĺ����ķ���ֵ������void����;�Դ����ͺ����ĵ������첽��,����
�豸��ȫ�����������֮ǰ�ͷ�����;�Դ����ͺ����ĵ��ñ���ָ��ִ������,��������
�豸��ִ�к���ʱ��grid��block��ά��,�Լ���ص���(������<<<   >>>�����);
a kernel,��ʾ�˺���Ϊ�ں˺���(������GPU�ϵ�CUDA���м��㺯����Ϊkernel(�ں˺�
��),�ں˺�������ͨ��__global__���������޶�������); */
__global__ static void green_ball(unsigned char* ptr, int width, int height)
{
	/* gridDim: ���ñ���,���������߳������ά��,���������߳̿���˵,���
	������һ������,���������̸߳�ÿһά�Ĵ�С,��ÿ���̸߳����߳̿������.
	Ϊdim3���ͣ�
	blockDim: ���ñ���,����˵��ÿ��block��ά����ߴ�.Ϊdim3����,����
	��block������ά���ϵĳߴ���Ϣ;���������߳̿���˵,���������һ������,
	��������߳̿���ÿһά���߳�����;
	blockIdx: ���ñ���,�����а�����ֵ���ǵ�ǰִ���豸������߳̿������;��
	��˵����ǰthread���ڵ�block������grid�е�λ��,blockIdx.xȡֵ��Χ��
	[0,gridDim.x-1],blockIdx.yȡֵ��Χ��[0, gridDim.y-1].Ϊuint3����,
	������һ��block��grid�и���ά���ϵ�������Ϣ;
	threadIdx: ���ñ���,�����а�����ֵ���ǵ�ǰִ���豸������߳�����;����
	˵����ǰthread��block�е�λ��;����߳���һά�Ŀɻ�ȡthreadIdx.x,���
	�Ƕ�ά�Ļ��ɻ�ȡthreadIdx.y,�������ά�Ļ��ɻ�ȡthreadIdx.z;Ϊuint3��
	��,������һ��thread��block�и���ά�ȵ�������Ϣ */
	// map from threadIdx/BlockIdx to pixel position
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	/* __shared__: ���������޶�����ʹ��__shared__�޶�����������__device__��
	�������ã���ʱ�����ı���λ��block�еĹ���洢���ռ��У���block������ͬ
	���������ڣ�����ͨ��block�ڵ������̷߳��ʣ�__shared__��__constant__����
	Ĭ��Ϊ�Ǿ�̬�洢����__shared__ǰ���Լ�extern�ؼ��֣�����ʾ���Ǳ�����С
	��ִ�в���ȷ����__shared__����������ʱ���ܳ�ʼ�������Խ�CUDA C�Ĺؼ���
	__shared__��ӵ����������У��⽫ʹ�������פ���ڹ����ڴ��У�CUDA C����
	���Թ����ڴ��еı�������ͨ�������ֱ��ȡ��ͬ�Ĵ���ʽ */
	__shared__ float shared[16][16]; // == threads_block

	// now calculate the value at that position
	const float period = 128.0f;

	shared[threadIdx.x][threadIdx.y] = 255 * (sinf(x*2.0f*PI / period) + 1.0f) *(sinf(y*2.0f*PI / period) + 1.0f) / 4.0f;

	/* __syncthreads: ���߳̿��е��߳̽���ͬ����CUDA�ܹ���ȷ���������߳̿�
	�е�ÿ���̶߳�ִ����__syncthreads()������û���κ��߳���ִ��
	__syncthreads()֮���ָ��;��ͬһ��block�е��߳�ͨ������洢��(shared
	memory)�������ݣ���ͨ��դ��ͬ��(������kernel��������Ҫͬ����λ�õ���
	__syncthreads()����)��֤�̼߳��ܹ���ȷ�ع������ݣ�ʹ��clock()������ʱ��
	���ں˺�����Ҫ������һ�δ���Ŀ�ʼ�ͽ�����λ�÷ֱ����һ��clock()������
	���������¼���������ڵ���__syncthreads()������һ��block�е�����
	thread��Ҫ��ʱ������ͬ�ģ����ֻ��Ҫ��¼ÿ��blockִ����Ҫ��ʱ������ˣ�
	������Ҫ��¼ÿ��thread��ʱ�� */
	// removing this syncthreads shows graphically what happens
	// when it doesn't exist.this is an example of why we need it.
	__syncthreads();

	ptr[offset * 4 + 0] = 0;
	ptr[offset * 4 + 1] = shared[/*15 - */threadIdx.x][/*15 - */threadIdx.y];
	ptr[offset * 4 + 2] = 0;
	ptr[offset * 4 + 3] = 255;
}

int green_ball_gpu(unsigned char* ptr, int width, int height, float* elapsed_time)
{
	/* hipEvent_t: CUDA event types,�ṹ������, CUDA�¼�,���ڲ���GPU��ĳ
	�������ϻ��ѵ�ʱ��,CUDA�е��¼���������һ��GPUʱ���,����CUDA�¼�����
	GPU��ʵ�ֵ�,������ǲ����ڶ�ͬʱ�����豸�������������Ļ�ϴ����ʱ */
	hipEvent_t start, stop;
	// hipEventCreate: ����һ���¼�����,�첽����
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// hipEventRecord: ��¼һ���¼�,�첽����,start��¼��ʼʱ��
	hipEventRecord(start, 0);

	const size_t length{ width * height * 4 * sizeof(unsigned char) };
	unsigned char* dev{ nullptr };
	// hipMalloc: ���豸�˷����ڴ�
	hipMalloc(&dev, length);

	const int threads_block{ 16 };
	dim3 blocks(width / threads_block, height / threads_block);
	dim3 threads(threads_block, threads_block);
	/* <<< >>>: ΪCUDA����������,ָ���߳�������߳̿�ά�ȵ�,����ִ�в�
	����CUDA������������ʱϵͳ,����˵���ں˺����е��߳�����,�Լ��߳������
	��֯��;����������Щ���������Ǵ��ݸ��豸����Ĳ���,���Ǹ�������ʱ���
	�����豸����,���ݸ��豸���뱾��Ĳ����Ƿ���Բ�����д��ݵ�,�����׼�ĺ�
	������һ��;��ͬ�����������豸���̵߳���������֯��ʽ�в�ͬ��Լ��;����
	��Ϊkernel���õ�����������������㹻�Ŀռ�,�ٵ���kernel����,������
	GPU����ʱ�ᷢ������,����Խ���;
	ʹ������ʱAPIʱ,��Ҫ�ڵ��õ��ں˺�����������б�ֱ����<<<Dg,Db,Ns,S>>>
	����ʽ����ִ������,���У�Dg��һ��dim3�ͱ���,��������grid��ά�Ⱥ͸���
	ά���ϵĳߴ�.���ú�Dg��,grid�н���Dg.x*Dg.y*Dg.z��block;Db��
	һ��dim3�ͱ���,��������block��ά�Ⱥ͸���ά���ϵĳߴ�.���ú�Db��,ÿ��
	block�н���Db.x*Db.y*Db.z��thread;Ns��һ��size_t�ͱ���,ָ������Ϊ�˵�
	�ö�̬����Ĺ���洢����С,��Щ��̬����Ĵ洢���ɹ�����Ϊ�ⲿ����
	(extern __shared__)�������κα���ʹ��;Ns��һ����ѡ����,Ĭ��ֵΪ0;SΪ
	hipStream_t����,�����������ں˺�����������.S��һ����ѡ����,Ĭ��ֵ0. */
	green_ball << <blocks, threads >> >(dev, width, height);

	/* hipMemcpy: �������˺��豸�˿�������,�˺������ĸ���������������֮һ:
	(1). hipMemcpyHostToHost: �������ݴ������˵�������
	(2). hipMemcpyHostToDevice: �������ݴ������˵��豸��
	(3). hipMemcpyDeviceToHost: �������ݴ��豸�˵�������
	(4). hipMemcpyDeviceToDevice: �������ݴ��豸�˵��豸��
	(5). hipMemcpyDefault: ��ָ��ֵ�Զ��ƶϿ������ݷ���,��Ҫ֧��
	ͳһ����Ѱַ(CUDA6.0�����ϰ汾)
	hipMemcpy��������������ͬ���� */
	hipMemcpy(ptr, dev, length, hipMemcpyDeviceToHost);

	// hipFree: �ͷ��豸����hipMalloc����������ڴ�
	hipFree(dev);

	// hipEventRecord: ��¼һ���¼�,�첽����,stop��¼����ʱ��
	hipEventRecord(stop, 0);
	// hipEventSynchronize: �¼�ͬ��,�ȴ�һ���¼����,�첽����
	hipEventSynchronize(stop);
	// cudaEventElapseTime: ���������¼�֮�侭����ʱ��,��λΪ����,�첽����
	hipEventElapsedTime(elapsed_time, start, stop);
	// hipEventDestroy: �����¼�����,�첽����
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}