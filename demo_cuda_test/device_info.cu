#include "funset.hpp"
#include <iostream>
#include <hip/hip_runtime.h> // For the CUDA runtime routines (prefixed with "cuda_")
#include <>

/* reference:
	C:\ProgramData\NVIDIA Corporation\CUDA Samples\v8.0\1_Utilities\deviceQuery
*/ 
int get_device_info()
{
	int device_count{ 0 };
	// hipGetDeviceCount: ��ü��������豸������
	hipGetDeviceCount(&device_count);
	fprintf(stdout, "GPU�豸�������� %d\n", device_count);

	for (int dev = 0; dev < device_count; ++dev) {
		int driver_version{ 0 }, runtime_version{ 0 };

		/* hipSetDevice: ����GPUִ��ʱʹ�õ��豸��0��ʾ���������ĵ�һ
		���豸�ţ�����ж���豸������Ϊ0,1,2... */
		hipSetDevice(dev);

		/* hipDeviceProp_t: �豸���Խṹ��
		name: �豸���֣���GeForce 940MX
		totalGlobalMem�� �豸�Ͽ��õ�ȫ���ڴ�����(�ֽ�)
		sharedMemPerBlock: ÿһ���߳̿��Ͽ��õĹ����ڴ�����(�ֽ�)
		regsPerBlock: ÿһ���߳̿��Ͽ��õ�32λ�Ĵ�������
		warpSize�� һ���߳����������߳���������ʵ�������У��߳̿�ᱻ�ָ�ɸ�С���߳���(warp)��
		           �߳����е�ÿ���̶߳����ڲ�ͬ������ִ����ͬ������
		memPitch: ���ڴ濽������������pitch��(�ֽ�)
		maxThreadsPerBlock: ÿһ���߳̿���֧�ֵ�����߳�����
		maxThreadsDim[3]: ÿһ���߳̿��ÿ��ά�ȵ�����С(x,y,z)
		maxGridSize: ÿһ���̸߳��ÿ��ά�ȵ�����С(x,y,z)
		clockRate�� GPU���ʱ��Ƶ��(ǧ����)
		totalConstMem: �豸�Ͽ��õĳ����ڴ�����(�ֽ�)
		major: �豸�����������汾�ţ��豸���������İ汾������һ��GPU��CUDA���ܵ�֧�̶ֳ�
		minor: �豸���������ΰ汾��
		textureAlignment: �������Ҫ��
		deviceOverlap: GPU�Ƿ�֧���豸�ص�(Device Overlap)����,֧���豸�ص����ܵ�GPU�ܹ�
		               ��ִ��һ��CUDA C�˺�����ͬʱ���������豸������֮��ִ�и��ƵȲ���,
			       �ѷ�����ʹ��asyncEngineCount����
		multiProcessorCount: �豸�϶ദ����������
		kernelExecTimeoutEnabled: ָ��ִ�к˺���ʱ�Ƿ�������ʱ������
		integrated: �豸�Ƿ���һ������GPU
		canMapHostMemory: �豸�Ƿ�֧��ӳ�������ڴ棬����Ϊ�Ƿ�֧���㿽���ڴ���ж�����
		computeMode: CUDA�豸����ģʽ���ɲο�hipComputeMode
		maxTexture1D: һά����֧�ֵ�����С
		maxTexture2D[2]����ά����֧�ֵ�����С(x,y)
		maxTexture3D[3]: ��ά����֧�ֵ�����С(x,y,z)
		memoryClockRate: �ڴ�ʱ��Ƶ�ʷ�ֵ(ǧ����)
		memoryBusWidth: ȫ���ڴ����߿��(bits)
		l2CacheSize: L2�����С(�ֽ�)
		maxThreadsPerMultiProcessor�� ÿ���ദ����֧�ֵ�����߳�����
		concurrentKernels: �豸�Ƿ�֧��ͬʱִ�ж���˺���
		asyncEngineCount: �첽��������
		unifiedAddressing: �Ƿ�֧���豸����������һ��ͳһ�ĵ�ַ�ռ�
		*/
		hipDeviceProp_t device_prop;
		/* hipGetDeviceProperties: ��ȡָ����GPU�豸���������Ϣ */
		hipGetDeviceProperties(&device_prop, dev);

		fprintf(stdout, "\n�豸 %d ����: %s\n", dev, device_prop.name);

		/* hipDriverGetVersion: ��ȡCUDA�����汾 */
		hipDriverGetVersion(&driver_version);
		fprintf(stdout, "CUDA�����汾�� %d.%d\n", driver_version/1000, (driver_version%1000)/10);
		/* hipRuntimeGetVersion: ��ȡCUDA����ʱ�汾 */
		hipRuntimeGetVersion(&runtime_version);
		fprintf(stdout, "CUDA����ʱ�汾�� %d.%d\n", runtime_version/1000, (runtime_version%1000)/10);

		fprintf(stdout, "�豸���������� %d.%d\n", device_prop.major, device_prop.minor);
		fprintf(stdout, "�豸�Ͽ��õ�ȫ���ڴ������� %f MB, %llu bytes\n",
			(float)device_prop.totalGlobalMem / (1024 * 1024), (unsigned long long)device_prop.totalGlobalMem);
		fprintf(stdout, "ÿһ���߳̿��Ͽ��õĹ����ڴ������� %f KB, %lu bytes\n",
			(float)device_prop.sharedMemPerBlock / 1024, device_prop.sharedMemPerBlock);
		fprintf(stdout, "ÿһ���߳̿��Ͽ��õ�32λ�Ĵ�������: %d\n", device_prop.regsPerBlock);
		fprintf(stdout, "һ���߳����������߳������� %d\n", device_prop.warpSize);
		fprintf(stdout, "���ڴ濽������������pitch��: %d bytes\n", device_prop.memPitch);
		fprintf(stdout, "ÿһ���߳̿���֧�ֵ�����߳�����: %d\n", device_prop.maxThreadsPerBlock);
		fprintf(stdout, "ÿһ���߳̿��ÿ��ά�ȵ�����С(x,y,z): (%d, %d, %d)\n",
			device_prop.maxThreadsDim[0], device_prop.maxThreadsDim[1], device_prop.maxThreadsDim[2]);
		fprintf(stdout, "ÿһ���̸߳��ÿ��ά�ȵ�����С(x,y,z): (%d, %d, %d)\n",
			device_prop.maxGridSize[0], device_prop.maxGridSize[1], device_prop.maxGridSize[2]);
		fprintf(stdout, "GPU���ʱ��Ƶ��: %.0f MHz (%0.2f GHz)\n",
			device_prop.clockRate*1e-3f, device_prop.clockRate*1e-6f);
		fprintf(stdout, "�豸�Ͽ��õĳ����ڴ�����: %lu bytes\n", device_prop.totalConstMem);
		fprintf(stdout, "�������Ҫ��: %lu bytes\n", device_prop.textureAlignment);
		fprintf(stdout, "�Ƿ�֧���豸�ص�����: %s\n", device_prop.deviceOverlap ? "Yes" : "No");
		fprintf(stdout, "�豸�϶ദ����������: %d\n", device_prop.multiProcessorCount);
		fprintf(stdout, "ִ�к˺���ʱ�Ƿ�������ʱ������: %s\n", device_prop.kernelExecTimeoutEnabled ? "Yes" : "No");
		fprintf(stdout, "�豸�Ƿ���һ������GPU: %s\n", device_prop.integrated ? "Yes" : "No");
		fprintf(stdout, "�豸�Ƿ�֧��ӳ�������ڴ�: %s\n", device_prop.canMapHostMemory ? "Yes" : "No");
		fprintf(stdout, "CUDA�豸����ģʽ: %d\n", device_prop.computeMode);
		fprintf(stdout, "һά����֧�ֵ�����С: %d\n", device_prop.maxTexture1D);
		fprintf(stdout, "��ά����֧�ֵ�����С(x,y): (%d, %d)\n", device_prop.maxTexture2D[0], device_prop.maxSurface2D[1]);
		fprintf(stdout, "��ά����֧�ֵ�����С(x,y,z): (%d, %d, %d)\n",
			device_prop.maxTexture3D[0], device_prop.maxSurface3D[1], device_prop.maxSurface3D[2]);
		fprintf(stdout, "�ڴ�ʱ��Ƶ�ʷ�ֵ: %.0f Mhz\n", device_prop.memoryClockRate * 1e-3f);
		fprintf(stdout, "ȫ���ڴ����߿��: %d bits\n", device_prop.memoryBusWidth);
		fprintf(stdout, "L2�����С: %d bytes\n", device_prop.l2CacheSize);
		fprintf(stdout, "ÿ���ദ����֧�ֵ�����߳�����: %d\n", device_prop.maxThreadsPerMultiProcessor);
		fprintf(stdout, "�豸�Ƿ�֧��ͬʱִ�ж���˺���: %s\n", device_prop.concurrentKernels ? "Yes" : "No");
		fprintf(stdout, "�첽��������: %d\n", device_prop.asyncEngineCount);
		fprintf(stdout, "�Ƿ�֧���豸����������һ��ͳһ�ĵ�ַ�ռ�: %s\n", device_prop.unifiedAddressing ? "Yes" : "No");
	}

	return 0;
}
