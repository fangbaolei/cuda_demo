#include "hip/hip_runtime.h"
#include "funset.hpp"
#include <iostream>
#include <hip/hip_runtime.h> // For the CUDA runtime routines (prefixed with "cuda_")
#include <>
#include "common.hpp"

// reference: C:\ProgramData\NVIDIA Corporation\CUDA Samples\v8.0\0_Simple\matrixMul
/* __global__: ���������޶���;���豸������;�������˵���,��������3.2�����Ͽ�����
�豸�˵���;�����ĺ����ķ���ֵ������void����;�Դ����ͺ����ĵ������첽��,����
�豸��ȫ�����������֮ǰ�ͷ�����;�Դ����ͺ����ĵ��ñ���ָ��ִ������,��������
�豸��ִ�к���ʱ��grid��block��ά��,�Լ���ص���(������<<<   >>>�����);
a kernel,��ʾ�˺���Ϊ�ں˺���(������GPU�ϵ�CUDA���м��㺯����Ϊkernel(�ں˺�
��),�ں˺�������ͨ��__global__���������޶�������);*/
template <int BLOCK_SIZE>
__global__ static void matrix_mul(const float* A, const float* B, float* C, int wA, int wB)
{
	/* gridDim: ���ñ���,���������߳������ά��,���������߳̿���˵,���
	������һ������,���������̸߳�ÿһά�Ĵ�С,��ÿ���̸߳����߳̿������.
	һ��gridΪ��ά,Ϊdim3���ͣ�
	blockDim: ���ñ���,����˵��ÿ��block��ά����ߴ�.Ϊdim3����,����
	��block������ά���ϵĳߴ���Ϣ;���������߳̿���˵,���������һ������,
	��������߳̿���ÿһά���߳�����;
	blockIdx: ���ñ���,�����а�����ֵ���ǵ�ǰִ���豸������߳̿������;��
	��˵����ǰthread���ڵ�block������grid�е�λ��,blockIdx.xȡֵ��Χ��
	[0,gridDim.x-1],blockIdx.yȡֵ��Χ��[0, gridDim.y-1].Ϊuint3����,
	������һ��block��grid�и���ά���ϵ�������Ϣ;
	threadIdx: ���ñ���,�����а�����ֵ���ǵ�ǰִ���豸������߳�����;����
	˵����ǰthread��block�е�λ��;����߳���һά�Ŀɻ�ȡthreadIdx.x,���
	�Ƕ�ά�Ļ��ɻ�ȡthreadIdx.y,�������ά�Ļ��ɻ�ȡthreadIdx.z;Ϊuint3��
	��,������һ��thread��block�и���ά�ȵ�������Ϣ */
	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;
	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	// Index of the first sub-matrix of A processed by the block
	int aBegin = wA * BLOCK_SIZE * by;
	// Index of the last sub-matrix of A processed by the block
	int aEnd = aBegin + wA - 1;
	// Step size used to iterate through the sub-matrices of A
	int aStep = BLOCK_SIZE;
	// Index of the first sub-matrix of B processed by the block
	int bBegin = BLOCK_SIZE * bx;
	// Step size used to iterate through the sub-matrices of B
	int bStep = BLOCK_SIZE * wB;
	// Csub is used to store the element of the block sub-matrix that is computed by the thread
	float Csub = 0;

	// Loop over all the sub-matrices of A and B required to compute the block sub-matrix
	for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
		/* __shared__: ���������޶�����ʹ��__shared__�޶�����������__device__��
		�������ã���ʱ�����ı���λ��block�еĹ���洢���ռ��У���block������ͬ
		���������ڣ�����ͨ��block�ڵ������̷߳��ʣ�__shared__��__constant__����
		Ĭ��Ϊ�Ǿ�̬�洢����__shared__ǰ���Լ�extern�ؼ��֣�����ʾ���Ǳ�����С
		��ִ�в���ȷ����__shared__����������ʱ���ܳ�ʼ�������Խ�CUDA C�Ĺؼ���
		__shared__��ӵ����������У��⽫ʹ�������פ���ڹ����ڴ��У�CUDA C����
		���Թ����ڴ��еı�������ͨ�������ֱ��ȡ��ͬ�Ĵ���ʽ */
		// Declaration of the shared memory array As used to store the sub-matrix of A
		__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
		// Declaration of the shared memory array Bs used to store the sub-matrix of B
		__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

		// Load the matrices from device memory to shared memory; each thread loads one element of each matrix
		As[ty][tx] = A[a + wA * ty + tx];
		Bs[ty][tx] = B[b + wB * ty + tx];

		/* __syncthreads: ���߳̿��е��߳̽���ͬ����CUDA�ܹ���ȷ���������߳̿�
		�е�ÿ���̶߳�ִ����__syncthreads()������û���κ��߳���ִ��
		__syncthreads()֮���ָ��;��ͬһ��block�е��߳�ͨ������洢��(shared
		memory)�������ݣ���ͨ��դ��ͬ��(������kernel��������Ҫͬ����λ�õ���
		__syncthreads()����)��֤�̼߳��ܹ���ȷ�ع������ݣ�ʹ��clock()������ʱ��
		���ں˺�����Ҫ������һ�δ���Ŀ�ʼ�ͽ�����λ�÷ֱ����һ��clock()������
		���������¼���������ڵ���__syncthreads()������һ��block�е�����
		thread��Ҫ��ʱ������ͬ�ģ����ֻ��Ҫ��¼ÿ��blockִ����Ҫ��ʱ������ˣ�
		������Ҫ��¼ÿ��thread��ʱ�� */
		// Synchronize to make sure the matrices are loaded
		__syncthreads();

		/* reference:
			https://devblogs.nvidia.com/parallelforall/new-compiler-features-cuda-8/
			https://stackoverflow.com/questions/22278631/what-does-pragma-unroll-do-exactly-does-it-affect-the-number-of-threads/22279341
		������Ĭ������½�ѭ��չ��С�Ĵ�����#pragma unroll�ܹ�ָ��ѭ��
		�Զ��ٴ�չ��(����Ա���뱣֤�����չ������ȷ��)��pragma unroll ��
		��������Ŵ����ѭ������ѡ��������һ�����֣�ָ������չ�����ٴ�ѭ����
		#pragma unroll 1 ��ʾ��ֹ��������ѭ��չ�������ûָ�����������ڳ���
		�ε�ѭ����ѭ������ȫչ�������ڲ�ȷ��������ѭ����ѭ��������չ����
		*/
#pragma unroll
		// Multiply the two matrices together; each thread computes one element of the block sub-matrix
		for (int k = 0; k < BLOCK_SIZE; ++k) {
			Csub += As[ty][k] * Bs[k][tx];
		}

		// Synchronize to make sure that the preceding computation is done before loading two new
		// sub-matrices of A and B in the next iteration
		__syncthreads();
	}

	// Write the block sub-matrix to device memory; each thread writes one element
	int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
	C[c + wB * ty + tx] = Csub;
}

__global__ static void matrix_mul(const float* A, const float* B, float* C, int colsA, int rowsA, int colsB, int rowsB)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	float sum{ 0.f };
	for (int t = 0; t < colsA; ++t) {
		sum += A[y * colsA + t] * B[t * colsB + x];
	}

	C[offset] = sum;
}

int matrix_mul_gpu(const float* A, const float* B, float* C, int colsA, int rowsA, int colsB, int rowsB, float* elapsed_time)
{
	CHECK(colsA == rowsB);

	/* hipEvent_t: CUDA event types���ṹ������, CUDA�¼������ڲ���GPU��ĳ
	�������ϻ��ѵ�ʱ�䣬CUDA�е��¼���������һ��GPUʱ���������CUDA�¼�����
	GPU��ʵ�ֵģ�������ǲ����ڶ�ͬʱ�����豸�������������Ļ�ϴ����ʱ*/
	hipEvent_t start, stop;
	// hipEventCreate: ����һ���¼������첽����
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// hipEventRecord: ��¼һ���¼����첽����,start��¼��ʼʱ��
	hipEventRecord(start, 0);

	size_t lengthA{ colsA * rowsA * sizeof(float) }, lengthB{ colsB * rowsB * sizeof(float) };
	size_t lengthC{ rowsA * colsB * sizeof(float) };
	float *d_A{ nullptr }, *d_B{ nullptr }, *d_C{ nullptr };

	// hipMalloc: ���豸�˷����ڴ�
	hipMalloc(&d_A, lengthA);
	hipMalloc(&d_B, lengthB);
	hipMalloc(&d_C, lengthC);

	/* hipMemcpy: �������˺��豸�˿�������,�˺������ĸ���������������֮һ:
	(1). hipMemcpyHostToHost: �������ݴ������˵�������
	(2). hipMemcpyHostToDevice: �������ݴ������˵��豸��
	(3). hipMemcpyDeviceToHost: �������ݴ��豸�˵�������
	(4). hipMemcpyDeviceToDevice: �������ݴ��豸�˵��豸��
	(5). hipMemcpyDefault: ��ָ��ֵ�Զ��ƶϿ������ݷ���,��Ҫ֧��
	ͳһ����Ѱַ(CUDA6.0�����ϰ汾)
	hipMemcpy��������������ͬ���� */
	hipMemcpy(d_A, A, lengthA, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, lengthB, hipMemcpyHostToDevice);
	//hipMemcpy(d_C, C, lengthC, hipMemcpyHostToDevice);

	const int block_size{ 32 };
	/* dim3: ����uint3���������ʸ�����ͣ��൱����3��unsigned int������ɵ�
	�ṹ�壬�ɱ�ʾһ����ά���飬�ڶ���dim3���ͱ���ʱ������û�и�ֵ��Ԫ�ض�
	�ᱻ����Ĭ��ֵ1 */
	dim3 dimsA(colsA, rowsA, 1);
	dim3 dimsB(colsB, rowsB, 1);
	CHECK(dimsA.x == dimsB.y);
	//fprintf(stderr, "MatrixA(%d,%d), MatrixB(%d,%d)\n", dimsA.x, dimsA.y, dimsB.x, dimsB.y);

	dim3 threads(block_size, block_size);
	dim3 grid(dimsB.x / threads.x, dimsA.y / threads.y);

	/* <<< >>>: ΪCUDA����������,ָ���߳�������߳̿�ά�ȵ�,����ִ�в�
	����CUDA������������ʱϵͳ,����˵���ں˺����е��߳�����,�Լ��߳������
	��֯��;����������Щ���������Ǵ��ݸ��豸����Ĳ���,���Ǹ�������ʱ���
	�����豸����,���ݸ��豸���뱾��Ĳ����Ƿ���Բ�����д��ݵ�,�����׼�ĺ�
	������һ��;��ͬ�����������豸���̵߳���������֯��ʽ�в�ͬ��Լ��;����
	��Ϊkernel���õ�����������������㹻�Ŀռ�,�ٵ���kernel����,������
	GPU����ʱ�ᷢ������,����Խ���;
	ʹ������ʱAPIʱ,��Ҫ�ڵ��õ��ں˺�����������б�ֱ����<<<Dg,Db,Ns,S>>>
	����ʽ����ִ������,���У�Dg��һ��dim3�ͱ���,��������grid��ά�Ⱥ͸���
	ά���ϵĳߴ�.���ú�Dg��,grid�н���Dg.x*Dg.y*Dg.z��block;Db��
	һ��dim3�ͱ���,��������block��ά�Ⱥ͸���ά���ϵĳߴ�.���ú�Db��,ÿ��
	block�н���Db.x*Db.y*Db.z��thread;Ns��һ��size_t�ͱ���,ָ������Ϊ�˵�
	�ö�̬����Ĺ���洢����С,��Щ��̬����Ĵ洢���ɹ�����Ϊ�ⲿ����
	(extern __shared__)�������κα���ʹ��;Ns��һ����ѡ����,Ĭ��ֵΪ0;SΪ
	hipStream_t����,�����������ں˺�����������.S��һ����ѡ����,Ĭ��ֵ0. */
	matrix_mul<block_size> <<< grid, threads >>>(d_A, d_B, d_C, dimsA.x, dimsB.x); // ���нϿ�
	//matrix_mul<< < grid, threads >> >(d_A, d_B, d_C, colsA, rowsA, colsB, rowsB);

	/* hipDeviceSynchronize: kernel���������첽��, Ϊ�˶�λ���Ƿ����, һ
	����Ҫ����hipDeviceSynchronize��������ͬ��; ����һֱ��������״̬��ֱ��
	ǰ����������������Ѿ���ȫ��ִ����ϣ����ǰ��ִ�е�ĳ������ʧ�ܣ�����
	����һ�����󣻵��������ж������������֮����ĳһ����Ҫͨ��ʱ���Ǿͱ���
	����һ�㴦����ͬ������䣬��hipDeviceSynchronize���첽����
	reference: https://stackoverflow.com/questions/11888772/when-to-call-cudadevicesynchronize */
	//hipDeviceSynchronize();

	hipMemcpy(C, d_C, lengthC, hipMemcpyDeviceToHost);
	// hipFree: �ͷ��豸����hipMalloc����������ڴ�
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	// hipEventRecord: ��¼һ���¼����첽����,stop��¼����ʱ��
	hipEventRecord(stop, 0);
	// hipEventSynchronize: �¼�ͬ�����ȴ�һ���¼���ɣ��첽����
	hipEventSynchronize(stop);
	// cudaEventElapseTime: ���������¼�֮�侭����ʱ�䣬��λΪ���룬�첽����
	hipEventElapsedTime(elapsed_time, start, stop);
	// hipEventDestroy: �����¼������첽����
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}

