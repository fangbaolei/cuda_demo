#include "hip/hip_runtime.h"
#include "funset.hpp"
#include <iostream>
#include <algorithm>
#include <memory>
#include <vector>
#include <hip/hip_runtime.h> // For the CUDA runtime routines (prefixed with "cuda_")
#include <>
#include "common.hpp"

/* __global__: ���������޶���;���豸������;�������˵���,��������3.2�����Ͽ�����
�豸�˵���;�����ĺ����ķ���ֵ������void����;�Դ����ͺ����ĵ������첽��,����
�豸��ȫ�����������֮ǰ�ͷ�����;�Դ����ͺ����ĵ��ñ���ָ��ִ������,��������
�豸��ִ�к���ʱ��grid��block��ά��,�Լ���ص���(������<<<   >>>�����);
a kernel,��ʾ�˺���Ϊ�ں˺���(������GPU�ϵ�CUDA���м��㺯����Ϊkernel(�ں˺�
��),�ں˺�������ͨ��__global__���������޶�������); */
__global__ static void stream_kernel(int* a, int* b, int* c, int length)
{
	/* gridDim: ���ñ���,���������߳������ά��,���������߳̿���˵,���
	������һ������,���������̸߳�ÿһά�Ĵ�С,��ÿ���̸߳����߳̿������.
	һ��gridΪ��ά,Ϊdim3���ͣ�
	blockDim: ���ñ���,����˵��ÿ��block��ά����ߴ�.Ϊdim3����,����
	��block������ά���ϵĳߴ���Ϣ;���������߳̿���˵,���������һ������,
	��������߳̿���ÿһά���߳�����;
	blockIdx: ���ñ���,�����а�����ֵ���ǵ�ǰִ���豸������߳̿������;��
	��˵����ǰthread���ڵ�block������grid�е�λ��,blockIdx.xȡֵ��Χ��
	[0,gridDim.x-1],blockIdx.yȡֵ��Χ��[0, gridDim.y-1].Ϊuint3����,
	������һ��block��grid�и���ά���ϵ�������Ϣ;
	threadIdx: ���ñ���,�����а�����ֵ���ǵ�ǰִ���豸������߳�����;����
	˵����ǰthread��block�е�λ��;����߳���һά�Ŀɻ�ȡthreadIdx.x,���
	�Ƕ�ά�Ļ��ɻ�ȡthreadIdx.y,�������ά�Ļ��ɻ�ȡthreadIdx.z;Ϊuint3��
	��,������һ��thread��block�и���ά�ȵ�������Ϣ */
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < length) {
		int idx1 = (idx + 1) % 256;
		int idx2 = (idx + 2) % 256;
		float as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
		float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
		c[idx] = (as + bs) / 2;
	}
}

int streams_gpu_1(const int* a, const int* b, int* c, int length, float* elapsed_time)
{
	// hipDeviceProp_t: cuda�豸���Խṹ��
	hipDeviceProp_t prop;
	// hipGetDeviceProperties: ��ȡGPU�豸�����Ϣ
	hipGetDeviceProperties(&prop, 0);
	/* hipDeviceProp_t::deviceOverlap: GPU�Ƿ�֧���豸�ص�(Device Overlap)��
	��,֧���豸�ص����ܵ�GPU�ܹ���ִ��һ��CUDA C�˺�����ͬʱ���������豸��
	����֮��ִ�и��ƵȲ��� */
	if (!prop.deviceOverlap) {
		printf("Device will not handle overlaps, so no speed up from streams\n");
		return -1;
	}

	/* hipEvent_t: CUDA event types,�ṹ������, CUDA�¼�,���ڲ���GPU��ĳ
	�������ϻ��ѵ�ʱ��,CUDA�е��¼���������һ��GPUʱ���,����CUDA�¼�����
	GPU��ʵ�ֵ�,������ǲ����ڶ�ͬʱ�����豸�������������Ļ�ϴ����ʱ */
	hipEvent_t start, stop;
	// hipEventCreate: ����һ���¼�����,�첽����
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// hipEventRecord: ��¼һ���¼�,�첽����,start��¼��ʼʱ��
	hipEventRecord(start, 0);

	/* hipStream_t: cuda �����ṹ������, CUDA����ʾһ��GPU�������У����Ҹ�
	�����еĲ�������ָ����˳��ִ�С����Խ�ÿ������ΪGPU�ϵ�һ�����񣬲�����
	Щ������Բ���ִ�С� */
	hipStream_t stream;
	// hipStreamCreate: ��ʼ����������һ���µ��첽��
	hipStreamCreate(&stream);

	int *host_a{ nullptr }, *host_b{ nullptr }, *host_c{ nullptr };
	int *dev_a{ nullptr }, *dev_b{ nullptr }, *dev_c{ nullptr };
	const int N{ length / 20 };

	// hipMalloc: ���豸�˷����ڴ�
	hipMalloc(&dev_a, N * sizeof(int));
	hipMalloc(&dev_b, N * sizeof(int));
	hipMalloc(&dev_c, N * sizeof(int));
	/* hipHostAlloc: ���������ڴ�(�̶��ڴ�)��C�⺯��malloc�������׼�ģ���
	��ҳ��(Pagable)�����ڴ棬��hipHostAlloc������ҳ�����������ڴ档ҳ������
	��Ҳ��Ϊ�̶��ڴ�(Pinned Memory)���߲��ɷ�ҳ�ڴ棬����һ����Ҫ�����ԣ�����ϵ
	ͳ�����������ڴ��ҳ�������������ϣ��Ӷ�ȷ���˸��ڴ�ʼ��פ����������
	���С���ˣ�����ϵͳ�ܹ���ȫ��ʹĳ��Ӧ�ó�����ʸ��ڴ�������ַ����Ϊ
	����ڴ潫���ᱻ�ƻ��������¶�λ������GPU֪���ڴ�������ַ����˿���ͨ
	��"ֱ���ڴ����(Direct Memory Access, DMA)"��������GPU������֮�临�����ݡ�
	�̶��ڴ���һ��˫�н�����ʹ�ù̶��ڴ�ʱ���㽫ʧȥ�����ڴ�����й��ܡ�
	���飺����hipMemcpy�����е�Դ�ڴ����Ŀ���ڴ棬��ʹ��ҳ�����ڴ棬������
	������Ҫʹ������ʱ�����ͷš� */
	// ��������ʹ�õ�ҳ�����ڴ�
	hipHostAlloc(&host_a, length * sizeof(int), hipHostMallocDefault);
	hipHostAlloc(&host_b, length * sizeof(int), hipHostMallocDefault);
	hipHostAlloc(&host_c, length * sizeof(int), hipHostMallocDefault);

	//for (int i = 0; i < length; ++i) {
	//	host_a[i] = a[i];
	//	host_b[i] = b[i];
	//}
	memcpy(host_a, a, length * sizeof(int));
	memcpy(host_b, b, length * sizeof(int));

	for (int i = 0; i < length; i += N) {
		/* hipMemcpyAsync: ��GPU������֮�临�����ݡ�hipMemcpy����Ϊ��
		����C�⺯��memcpy�������ǣ������������ͬ����ʽִ�У�����ζ�ţ�
		����������ʱ�����Ʋ������Ѿ���ɣ�����������������а����˸���
		��ȥ�����ݡ��첽��������Ϊ��ͬ�������෴���ڵ���hipMemcpyAsyncʱ��
		ֻ�Ƿ�����һ�����󣬱�ʾ������ִ��һ���ڴ渴�Ʋ������������ͨ��
		����stream��ָ���ġ�����������ʱ�������޷�ȷ�����Ʋ����Ƿ��Ѿ�
		���������޷���֤�����Ƿ��Ѿ������������ܹ��õ��ı�֤�ǣ����Ʋ����϶�
		�ᵱ��һ�����������еĲ���֮ǰִ�С��κδ��ݸ�hipMemcpyAsync������
		�ڴ�ָ�붼�����Ѿ�ͨ��hipHostAlloc������ڴ档Ҳ���ǣ���ֻ�����첽
		��ʽ��ҳ�����ڴ���и��Ʋ��� */
		// �������ڴ����첽��ʽ���Ƶ��豸��
		hipMemcpyAsync(dev_a, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream);
		hipMemcpyAsync(dev_b, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream);

		/* <<< >>>: ΪCUDA����������,ָ���߳�������߳̿�ά�ȵ�,����ִ�в�
		����CUDA������������ʱϵͳ,����˵���ں˺����е��߳�����,�Լ��߳������
		��֯��;����������Щ���������Ǵ��ݸ��豸����Ĳ���,���Ǹ�������ʱ���
		�����豸����,���ݸ��豸���뱾��Ĳ����Ƿ���Բ�����д��ݵ�,�����׼�ĺ�
		������һ��;��ͬ�����������豸���̵߳���������֯��ʽ�в�ͬ��Լ��;����
		��Ϊkernel���õ�����������������㹻�Ŀռ�,�ٵ���kernel����,������
		GPU����ʱ�ᷢ������,����Խ���;
		ʹ������ʱAPIʱ,��Ҫ�ڵ��õ��ں˺�����������б�ֱ����<<<Dg,Db,Ns,S>>>
		����ʽ����ִ������,���У�Dg��һ��dim3�ͱ���,��������grid��ά�Ⱥ͸���
		ά���ϵĳߴ�.���ú�Dg��,grid�н���Dg.x*Dg.y*Dg.z��block;Db��
		һ��dim3�ͱ���,��������block��ά�Ⱥ͸���ά���ϵĳߴ�.���ú�Db��,ÿ��
		block�н���Db.x*Db.y*Db.z��thread;Ns��һ��size_t�ͱ���,ָ������Ϊ�˵�
		�ö�̬����Ĺ���洢����С,��Щ��̬����Ĵ洢���ɹ�����Ϊ�ⲿ����
		(extern __shared__)�������κα���ʹ��;Ns��һ����ѡ����,Ĭ��ֵΪ0;SΪ
		hipStream_t����,�����������ں˺�����������.S��һ����ѡ����,Ĭ��ֵ0. */
		stream_kernel << <N / 256, 256, 0, stream >> >(dev_a, dev_b, dev_c, N);

		hipMemcpyAsync(host_c + i, dev_c, N * sizeof(int), hipMemcpyDeviceToHost, stream);
	}

	/* hipStreamSynchronize: �ȴ��������еĲ�����ɣ������ڼ���ִ��֮ǰ��Ҫ
	�ȴ�GPUִ����� */
	hipStreamSynchronize(stream);

	//for (int i = 0; i < length; ++i)
	//	c[i] = host_c[i];
	memcpy(c, host_c, length * sizeof(int));

	// hipHostFree: �ͷ��豸����hipHostAlloc����������ڴ�
	hipHostFree(host_a);
	hipHostFree(host_b);
	hipHostFree(host_c);
	// hipFree: �ͷ��豸����hipMalloc����������ڴ�
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	// hipStreamDestroy: ������
	hipStreamDestroy(stream);

	// hipEventRecord: ��¼һ���¼�,�첽����,stop��¼����ʱ��
	hipEventRecord(stop, 0);
	// hipEventSynchronize: �¼�ͬ��,�ȴ�һ���¼����,�첽����
	hipEventSynchronize(stop);
	// cudaEventElapseTime: ���������¼�֮�侭����ʱ��,��λΪ����,�첽����
	hipEventElapsedTime(elapsed_time, start, stop);
	// hipEventDestroy: �����¼�����,�첽����
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}

int streams_gpu_2(const int* a, const int* b, int* c, int length, float* elapsed_time)
{
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	if (!prop.deviceOverlap) {
		printf("Device will not handle overlaps, so no speed up from streams\n");
		return -1;
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	hipStream_t stream0, stream1;
	hipStreamCreate(&stream0);
	hipStreamCreate(&stream1);

	int *host_a{ nullptr }, *host_b{ nullptr }, *host_c{ nullptr };
	int *dev_a0{ nullptr }, *dev_b0{ nullptr }, *dev_c0{ nullptr };
	int *dev_a1{ nullptr }, *dev_b1{ nullptr }, *dev_c1{ nullptr };
	const int N{ length / 20 };

	hipMalloc(&dev_a0, N * sizeof(int));
	hipMalloc(&dev_b0, N * sizeof(int));
	hipMalloc(&dev_c0, N * sizeof(int));
	hipMalloc(&dev_a1, N * sizeof(int));
	hipMalloc(&dev_b1, N * sizeof(int));
	hipMalloc(&dev_c1, N * sizeof(int));
	hipHostAlloc(&host_a, length * sizeof(int), hipHostMallocDefault);
	hipHostAlloc(&host_b, length * sizeof(int), hipHostMallocDefault);
	hipHostAlloc(&host_c, length * sizeof(int), hipHostMallocDefault);

	memcpy(host_a, a, length * sizeof(int));
	memcpy(host_b, b, length * sizeof(int));

	for (int i = 0; i < length; i += N * 2) {
		//hipMemcpyAsync(dev_a0, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream0);
		//hipMemcpyAsync(dev_b0, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream0);
		//stream_kernel << <N / 256, 256, 0, stream0 >> >(dev_a0, dev_b0, dev_c0, N);
		//hipMemcpyAsync(host_c + i, dev_c0, N * sizeof(int), hipMemcpyDeviceToHost, stream0);

		//hipMemcpyAsync(dev_a1, host_a + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1);
		//hipMemcpyAsync(dev_b1, host_b + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1);
		//stream_kernel << <N / 256, 256, 0, stream1 >> >(dev_a1, dev_b1, dev_c1, N);
		//hipMemcpyAsync(host_c + i + N, dev_c1, N * sizeof(int), hipMemcpyDeviceToHost, stream1);

		// �Ƽ����ÿ�����ȷ�ʽ
		hipMemcpyAsync(dev_a0, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream0);
		hipMemcpyAsync(dev_a1, host_a + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1);

		hipMemcpyAsync(dev_b0, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream0);
		hipMemcpyAsync(dev_b1, host_b + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1);

		stream_kernel << <N / 256, 256, 0, stream0 >> >(dev_a0, dev_b0, dev_c0, N);
		stream_kernel << <N / 256, 256, 0, stream1 >> >(dev_a1, dev_b1, dev_c1, N);

		hipMemcpyAsync(host_c + i, dev_c0, N * sizeof(int), hipMemcpyDeviceToHost, stream0);
		hipMemcpyAsync(host_c + i + N, dev_c1, N * sizeof(int), hipMemcpyDeviceToHost, stream1);
	}

	hipStreamSynchronize(stream0);
	hipStreamSynchronize(stream1);

	memcpy(c, host_c, length * sizeof(int));

	hipHostFree(host_a);
	hipHostFree(host_b);
	hipHostFree(host_c);
	hipFree(dev_a0);
	hipFree(dev_b0);
	hipFree(dev_c0);
	hipFree(dev_a1);
	hipFree(dev_b1);
	hipFree(dev_c1);
	hipStreamDestroy(stream0);
	hipStreamDestroy(stream1);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(elapsed_time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}

int streams_gpu(const int* a, const int* b, int* c, int length, float* elapsed_time)
{
	int ret{ 0 };
	//ret = streams_gpu_1(a, b, c, length, elapsed_time); // ʹ�õ�����
	ret = streams_gpu_2(a, b, c, length, elapsed_time); // ʹ�ö����

	return ret;
}
