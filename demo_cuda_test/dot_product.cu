#include "hip/hip_runtime.h"
#include "funset.hpp"
#include <iostream>
#include <algorithm>
#include <memory>
#include <hip/hip_runtime.h> // For the CUDA runtime routines (prefixed with "cuda_")
#include <>
#include "common.hpp"

/* __global__: ���������޶���;���豸������;�������˵���,��������3.2�����Ͽ�����
�豸�˵���;�����ĺ����ķ���ֵ������void����;�Դ����ͺ����ĵ������첽��,����
�豸��ȫ�����������֮ǰ�ͷ�����;�Դ����ͺ����ĵ��ñ���ָ��ִ������,��������
�豸��ִ�к���ʱ��grid��block��ά��,�Լ���ص���(������<<<   >>>�����);
a kernel,��ʾ�˺���Ϊ�ں˺���(������GPU�ϵ�CUDA���м��㺯����Ϊkernel(�ں˺�
��),�ں˺�������ͨ��__global__���������޶�������);*/
__global__ static void dot_product(const float* A, const float* B, float* partial_C, int elements_num)
{
	/* __shared__: ���������޶�����ʹ��__shared__�޶�����������__device__��
	�������ã���ʱ�����ı���λ��block�еĹ���洢���ռ��У���block������ͬ
	���������ڣ�����ͨ��block�ڵ������̷߳��ʣ�__shared__��__constant__����
	Ĭ��Ϊ�Ǿ�̬�洢����__shared__ǰ���Լ�extern�ؼ��֣�����ʾ���Ǳ�����С
	��ִ�в���ȷ����__shared__����������ʱ���ܳ�ʼ�������Խ�CUDA C�Ĺؼ���
	__shared__��ӵ����������У��⽫ʹ�������פ���ڹ����ڴ��У�CUDA C����
	���Թ����ڴ��еı�������ͨ�������ֱ��ȡ��ͬ�Ĵ���ʽ */
	__shared__ float cache[256]; // == threadsPerBlock

	/* gridDim: ���ñ���,���������߳������ά��,���������߳̿���˵,���
	������һ������,���������̸߳�ÿһά�Ĵ�С,��ÿ���̸߳����߳̿������.
	Ϊdim3���ͣ�
	blockDim: ���ñ���,����˵��ÿ��block��ά����ߴ�.Ϊdim3����,����
	��block������ά���ϵĳߴ���Ϣ;���������߳̿���˵,���������һ������,
	��������߳̿���ÿһά���߳�����;
	blockIdx: ���ñ���,�����а�����ֵ���ǵ�ǰִ���豸������߳̿������;��
	��˵����ǰthread���ڵ�block������grid�е�λ��,blockIdx.xȡֵ��Χ��
	[0,gridDim.x-1],blockIdx.yȡֵ��Χ��[0, gridDim.y-1].Ϊuint3����,
	������һ��block��grid�и���ά���ϵ�������Ϣ;
	threadIdx: ���ñ���,�����а�����ֵ���ǵ�ǰִ���豸������߳�����;����
	˵����ǰthread��block�е�λ��;����߳���һά�Ŀɻ�ȡthreadIdx.x,���
	�Ƕ�ά�Ļ��ɻ�ȡthreadIdx.y,�������ά�Ļ��ɻ�ȡthreadIdx.z;Ϊuint3��
	��,������һ��thread��block�и���ά�ȵ�������Ϣ */
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int cacheIndex = threadIdx.x;

	float tmp{ 0.f };
	while (tid < elements_num) {
		tmp += A[tid] * B[tid];
		tid += blockDim.x * gridDim.x;
	}

	// ����cache����Ӧλ���ϵ�ֵ
	// �����ڴ滺���е�ƫ�ƾ͵����߳��������߳̿����������ƫ���޹أ���Ϊÿ
	// ���߳̿鶼ӵ�иù����ڴ��˽�и���
	cache[cacheIndex] = tmp;

	/* __syncthreads: ���߳̿��е��߳̽���ͬ����CUDA�ܹ���ȷ���������߳̿�
	�е�ÿ���̶߳�ִ����__syncthreads()������û���κ��߳���ִ��
	__syncthreads()֮���ָ��;��ͬһ��block�е��߳�ͨ������洢��(shared 
	memory)�������ݣ���ͨ��դ��ͬ��(������kernel��������Ҫͬ����λ�õ���
	__syncthreads()����)��֤�̼߳��ܹ���ȷ�ع������ݣ�ʹ��clock()������ʱ��
	���ں˺�����Ҫ������һ�δ���Ŀ�ʼ�ͽ�����λ�÷ֱ����һ��clock()������
	���������¼���������ڵ���__syncthreads()������һ��block�е�����
	thread��Ҫ��ʱ������ͬ�ģ����ֻ��Ҫ��¼ÿ��blockִ����Ҫ��ʱ������ˣ�
	������Ҫ��¼ÿ��thread��ʱ�� */
	__syncthreads();

	// ���ڹ�Լ������˵������codeҪ��threadPerBlock������2��ָ��
	int i = blockDim.x / 2;
	while (i != 0) {
		if (cacheIndex < i)
			cache[cacheIndex] += cache[cacheIndex + i];

		// ��ѭ�������и����˹����ڴ����cache��������ѭ������һ�ε�����ʼ֮ǰ��
		// ��Ҫȷ����ǰ�����������̵߳ĸ��²������Ѿ����
		__syncthreads();
		i /= 2;
	}

	// ֻ��cacheIndex == 0���߳�ִ��������������������Ϊֻ��һ��ֵд�뵽
	// ȫ���ڴ棬���ֻ��Ҫһ���߳���ִ�������������Ȼ��Ҳ����ѡ���κ�һ��
	// �߳̽�cache[0]д�뵽ȫ���ڴ�
	if (cacheIndex == 0)
		partial_C[blockIdx.x] = cache[0];
}

static int dot_product_gpu_1(const float* A, const float* B, float* value, int elements_num, float* elapsed_time)
{
	/* hipEvent_t: CUDA event types,�ṹ������, CUDA�¼�,���ڲ���GPU��ĳ
	�������ϻ��ѵ�ʱ��,CUDA�е��¼���������һ��GPUʱ���,����CUDA�¼�����
	GPU��ʵ�ֵ�,������ǲ����ڶ�ͬʱ�����豸�������������Ļ�ϴ����ʱ*/
	hipEvent_t start, stop;
	// hipEventCreate: ����һ���¼�����,�첽����
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// hipEventRecord: ��¼һ���¼�,�첽����,start��¼��ʼʱ��
	hipEventRecord(start, 0);

	size_t lengthA{ elements_num * sizeof(float) }, lengthB{ elements_num * sizeof(float) };
	float *d_A{ nullptr }, *d_B{ nullptr }, *d_partial_C{ nullptr };

	// hipMalloc: ���豸�˷����ڴ�
	hipMalloc(&d_A, lengthA);
	hipMalloc(&d_B, lengthB);

	/* hipMemcpy: �������˺��豸�˿�������,�˺������ĸ���������������֮һ:
	(1). hipMemcpyHostToHost: �������ݴ������˵�������
	(2). hipMemcpyHostToDevice: �������ݴ������˵��豸��
	(3). hipMemcpyDeviceToHost: �������ݴ��豸�˵�������
	(4). hipMemcpyDeviceToDevice: �������ݴ��豸�˵��豸��
	(5). hipMemcpyDefault: ��ָ��ֵ�Զ��ƶϿ������ݷ���,��Ҫ֧��
	ͳһ����Ѱַ(CUDA6.0�����ϰ汾)
	hipMemcpy��������������ͬ���� */
	hipMemcpy(d_A, A, lengthA, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, lengthB, hipMemcpyHostToDevice);

	const int threadsPerBlock{ 256 };
	const int blocksPerGrid = std::min(64, (elements_num + threadsPerBlock - 1) / threadsPerBlock);
	size_t lengthC{ blocksPerGrid * sizeof(float) };
	hipMalloc(&d_partial_C, lengthC);

	/* <<< >>>: ΪCUDA����������,ָ���߳�������߳̿�ά�ȵ�,����ִ�в�
	����CUDA������������ʱϵͳ,����˵���ں˺����е��߳�����,�Լ��߳������
	��֯��;����������Щ���������Ǵ��ݸ��豸����Ĳ���,���Ǹ�������ʱ���
	�����豸����,���ݸ��豸���뱾��Ĳ����Ƿ���Բ�����д��ݵ�,�����׼�ĺ�
	������һ��;��ͬ�����������豸���̵߳���������֯��ʽ�в�ͬ��Լ��;����
	��Ϊkernel���õ�����������������㹻�Ŀռ�,�ٵ���kernel����,������
	GPU����ʱ�ᷢ������,����Խ���;
	ʹ������ʱAPIʱ,��Ҫ�ڵ��õ��ں˺�����������б�ֱ����<<<Dg,Db,Ns,S>>>
	����ʽ����ִ������,���У�Dg��һ��dim3�ͱ���,��������grid��ά�Ⱥ͸���
	ά���ϵĳߴ�.���ú�Dg��,grid�н���Dg.x*Dg.y*Dg.z��block;Db��
	һ��dim3�ͱ���,��������block��ά�Ⱥ͸���ά���ϵĳߴ�.���ú�Db��,ÿ��
	block�н���Db.x*Db.y*Db.z��thread;Ns��һ��size_t�ͱ���,ָ������Ϊ�˵�
	�ö�̬����Ĺ���洢����С,��Щ��̬����Ĵ洢���ɹ�����Ϊ�ⲿ����
	(extern __shared__)�������κα���ʹ��;Ns��һ����ѡ����,Ĭ��ֵΪ0;SΪ
	hipStream_t����,�����������ں˺�����������.S��һ����ѡ����,Ĭ��ֵ0. */
	dot_product << < blocksPerGrid, threadsPerBlock >> >(d_A, d_B, d_partial_C, elements_num);

	/* hipDeviceSynchronize: kernel���������첽��, Ϊ�˶�λ���Ƿ����, һ
	����Ҫ����hipDeviceSynchronize��������ͬ��; ����һֱ��������״̬,ֱ��
	ǰ����������������Ѿ���ȫ��ִ�����,���ǰ��ִ�е�ĳ������ʧ��,����
	����һ�����󣻵��������ж����,������֮����ĳһ����Ҫͨ��ʱ,�Ǿͱ���
	����һ�㴦����ͬ�������,��hipDeviceSynchronize���첽����
	reference: https://stackoverflow.com/questions/11888772/when-to-call-cudadevicesynchronize */
	//hipDeviceSynchronize();

	std::unique_ptr<float[]> partial_C(new float[blocksPerGrid]);
	hipMemcpy(partial_C.get(), d_partial_C, lengthC, hipMemcpyDeviceToHost);

	*value = 0.f;
	for (int i = 0; i < blocksPerGrid; ++i) {
		(*value) += partial_C[i];
	}

	// hipFree: �ͷ��豸����hipMalloc����������ڴ�
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_partial_C);

	// hipEventRecord: ��¼һ���¼�,�첽����,stop��¼����ʱ��
	hipEventRecord(stop, 0);
	// hipEventSynchronize: �¼�ͬ��,�ȴ�һ���¼����,�첽����
	hipEventSynchronize(stop);
	// cudaEventElapseTime: ���������¼�֮�侭����ʱ��,��λΪ����,�첽����
	hipEventElapsedTime(elapsed_time, start, stop);
	// hipEventDestroy: �����¼�����,�첽����
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}

static int dot_product_gpu_2(const float* A, const float* B, float* value, int elements_num, float* elapsed_time)
{
	// hipDeviceProp_t: cuda�豸���Խṹ��
	hipDeviceProp_t prop;
	int count;
	// hipGetDeviceCount: ��ü��������豸������
	hipGetDeviceCount(&count);
	//fprintf(stderr, "device count: %d\n", count);
	int whichDevice;
	// hipGetDevice: ��õ�ǰ����ʹ�õ��豸ID���豸ID��0��ʼ���
	hipGetDevice(&whichDevice);
	// hipGetDeviceProperties: ��ȡGPU�豸�����Ϣ
	hipGetDeviceProperties(&prop, whichDevice);
	// hipDeviceProp_t::canMapHostMemory: GPU�Ƿ�֧���豸ӳ�������ڴ�
	if (prop.canMapHostMemory != 1) {
		fprintf(stderr, "Device cannot map memory.\n");
		return -1;
	}
	
	// hipSetDeviceFlags: �����豸Ҫ����ִ�еı�־
	// ���豸�����ܷ����㿽���ڴ��״̬
	hipSetDeviceFlags(hipDeviceMapHost);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	const int threadsPerBlock{ 256 };
	const int blocksPerGrid = std::min(64, (elements_num + threadsPerBlock - 1) / threadsPerBlock);

	size_t lengthA{ elements_num * sizeof(float) }, lengthB{ elements_num * sizeof(float) };
	float *d_A{ nullptr }, *d_B{ nullptr }, *d_partial_C{ nullptr };
	float *a{ nullptr }, *b{ nullptr }, *partial_c{ nullptr };

	/* hipHostAlloc: ���������ڴ档C�⺯��malloc�������׼�ģ���
	��ҳ��(Pagable)�����ڴ棬��hipHostAlloc������ҳ�����������ڴ档ҳ������
	��Ҳ��Ϊ�̶��ڴ�(Pinned Memory)���߲��ɷ�ҳ�ڴ棬����һ����Ҫ�����ԣ�����ϵ
	ͳ�����������ڴ��ҳ�������������ϣ��Ӷ�ȷ���˸��ڴ�ʼ��פ����������
	���С���ˣ�����ϵͳ�ܹ���ȫ��ʹĳ��Ӧ�ó�����ʸ��ڴ�������ַ����Ϊ
	����ڴ潫���ᱻ�ƻ��������¶�λ������GPU֪���ڴ�������ַ����˿���ͨ
	��"ֱ���ڴ����(Direct Memory Access, DMA)"��������GPU������֮�临�����ݡ�
	�̶��ڴ���һ��˫�н�����ʹ�ù̶��ڴ�ʱ���㽫ʧȥ�����ڴ�����й��ܡ�
	���飺����hipMemcpy�����е�Դ�ڴ����Ŀ���ڴ棬��ʹ��ҳ�����ڴ棬������
	������Ҫʹ������ʱ�����ͷš�
	�㿽���ڴ棺ͨ��hipHostAlloc����+hipHostMallocMapped���������̶��ڴ���
	hipHostAlloc����+hipHostMallocDefault������ͨ��hipHostMallocMapped����
	�������ڴ�Ҳ�ǹ̶��ģ�����ͨ��hipHostMallocDefault����Ĺ̶��ڴ�������ͬ
	�����ԡ��������ڴ���˿�������������GPU֮����ڴ渴���⣬��������CUDA C��
	������ֱ�ӷ����������͵������ڴ棬������Ҫ���Ƶ�GPU�����Ҳ��Ϊ�㿽���ڴ档
	hipHostMallocMapped�������־��������ʱ����GPU�з�������ڴ档
	hipHostMallocWriteCombined�������־��ʾ������ʱӦ�ý��ڴ����Ϊ"�ϲ�ʽд
	��(Write-Combined)"�ڴ档�����־������ı�Ӧ�ó�������ܣ���ȴ����������
	����GPU��ȡ�ڴ�ʱ�����ܡ�Ȼ������CPUҲҪ��ȡ����ڴ�ʱ��"�ϲ�ʽд��"���Ե�
	�ܵ�Ч��
	���ڼ���GPU��ʹ���㿽���ڴ�ͨ���������������������Ϊ�ڴ�����������������
	����ġ�������������Ϊ�㿽���ڴ��Ψһ���þ��Ǳ��ⲻ��Ҫ�����ݸ��ơ���������
	�Ĺ̶��ڴ涼����һ���ľ����ԣ��㿽���ڴ�ͬ�������⣺ÿ���̶��ڴ涼��ռ��ϵͳ
	�Ŀ��������ڴ棬�����ս�����ϵͳ�����ܡ�
	�������ڴ������ڴ涼ֻ��ʹ��һ��ʱ����ô�ڶ���GPU��ʹ���㿽���ڴ潫�������������� */
	// allocate the memory on the CPU
	hipHostAlloc(&a, lengthA, hipHostMallocWriteCombined | hipHostMallocMapped);
	hipHostAlloc(&b, lengthB, hipHostMallocWriteCombined | hipHostMallocMapped);
	hipHostAlloc(&partial_c, blocksPerGrid * sizeof(float), hipHostMallocMapped);

	/* hipHostGetDevicePointer: �����hipHostAlloc�����ӳ�������ڴ���豸ָ�롣
	����GPU�������ڴ�ռ��ַӳ����CPU��ͬ����hipHostAlloc���ص���CPU�ϵ�ָ�룬
	�����Ҫ����hipHostGetDevicePointer�������������ڴ���GPU�ϵ���Чָ�롣��Щָ��
	�������ݸ��˺��������������GPU������ڴ�ִ�ж�ȡ��д��Ȳ��� */
	// find out the GPU pointers
	hipHostGetDevicePointer(&d_A, a, 0);
	hipHostGetDevicePointer(&d_B, b, 0);
	hipHostGetDevicePointer(&d_partial_C, partial_c, 0);

	memcpy(a, A, lengthA);
	memcpy(b, B, lengthB);

	dot_product << < blocksPerGrid, threadsPerBlock >> >(d_A, d_B, d_partial_C, elements_num);

	/* hipDeviceSynchronize: �ȴ������豸���, ��CPU��GPUͬ��*/
	hipDeviceSynchronize();

	*value = 0.f;
	for (int i = 0; i < blocksPerGrid; ++i) {
		(*value) += partial_c[i];
	}

	// hipHostFree: �ͷ��豸����hipHostAlloc����������ڴ�
	hipHostFree(d_A);
	hipHostFree(d_B);
	hipHostFree(d_partial_C);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(elapsed_time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}

int dot_product_gpu(const float* A, const float* B, float* value, int elements_num, float* elapsed_time)
{
	int ret{ 0 };
	//ret = dot_product_gpu_1(A, B, value, elements_num, elapsed_time); // ��ͨʵ��
	ret = dot_product_gpu_2(A, B, value, elements_num, elapsed_time); // ͨ���㿽���ڴ�ʵ��

	return ret;
}
