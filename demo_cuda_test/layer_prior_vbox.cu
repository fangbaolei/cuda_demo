#include "hip/hip_runtime.h"
#include "funset.hpp"
#include <iostream>
#include <memory>
#include <algorithm>
#include <cmath>
#include <hip/hip_runtime.h> // For the CUDA runtime routines (prefixed with "cuda_")
#include <>
#include "common.hpp"

/* __global__: ���������޶���;���豸������;�������˵���,��������3.2�����Ͽ�����
�豸�˵���;�����ĺ����ķ���ֵ������void����;�Դ����ͺ����ĵ������첽��,����
�豸��ȫ�����������֮ǰ�ͷ�����;�Դ����ͺ����ĵ��ñ���ָ��ִ������,��������
�豸��ִ�к���ʱ��grid��block��ά��,�Լ���ص���(������<<<   >>>�����);
a kernel,��ʾ�˺���Ϊ�ں˺���(������GPU�ϵ�CUDA���м��㺯����Ϊkernel(�ں˺�
��),�ں˺�������ͨ��__global__���������޶�������);*/
__global__ static void layer_prior_vbox(float* dst, int layer_width, int layer_height, int image_width, int image_height,
	float offset, float step, int num_priors, float width, const float* height, const float* variance, int channel_size)
{
	/* gridDim: ���ñ���,���������߳������ά��,���������߳̿���˵,���
	������һ������,���������̸߳�ÿһά�Ĵ�С,��ÿ���̸߳����߳̿������.
	һ��gridΪ��ά,Ϊdim3���ͣ�
	blockDim: ���ñ���,����˵��ÿ��block��ά����ߴ�.Ϊdim3����,����
	��block������ά���ϵĳߴ���Ϣ;���������߳̿���˵,���������һ������,
	��������߳̿���ÿһά���߳�����;
	blockIdx: ���ñ���,�����а�����ֵ���ǵ�ǰִ���豸������߳̿������;��
	��˵����ǰthread���ڵ�block������grid�е�λ��,blockIdx.xȡֵ��Χ��
	[0,gridDim.x-1],blockIdx.yȡֵ��Χ��[0, gridDim.y-1].Ϊuint3����,
	������һ��block��grid�и���ά���ϵ�������Ϣ;
	threadIdx: ���ñ���,�����а�����ֵ���ǵ�ǰִ���豸������߳�����;����
	˵����ǰthread��block�е�λ��;����߳���һά�Ŀɻ�ȡthreadIdx.x,���
	�Ƕ�ά�Ļ��ɻ�ȡthreadIdx.y,�������ά�Ļ��ɻ�ȡthreadIdx.z;Ϊuint3��
	��,������һ��thread��block�и���ά�ȵ�������Ϣ */
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < layer_width && y < layer_height) {
		float center_x = (x + offset) * step;
		float center_y = (y + offset) * step;
		int idx = x * num_priors * 4 + y * (layer_width * num_priors * 4);

		for (int s = 0; s < num_priors; ++s) {
			float box_width = width;
			float box_height = height[s];
			int idx1 = idx + s * 4;

			dst[idx1] = (center_x - box_width / 2.) / image_width;
			dst[idx1 + 1] = (center_y - box_height / 2.) / image_height;
			dst[idx1 + 2] = (center_x + box_width / 2.) / image_width;
			dst[idx1 + 3] = (center_y + box_height / 2.) / image_height;

			int idx2 = channel_size + idx + s * 4;
			dst[idx2] = variance[0];
			dst[idx2 + 1] = variance[1];
			dst[idx2 + 2] = variance[2];
			dst[idx2 + 3] = variance[3];
		}
	}
}

int layer_prior_vbox_gpu(float* dst, int length, const std::vector<float>& vec1, const std::vector<float>& vec2,
	const std::vector<float>& vec3, float* elapsed_time)
{
	float *dev_dst{ nullptr }, *dev_vec;
	// hipMalloc: ���豸�˷����ڴ�
	hipMalloc(&dev_dst, length * sizeof(float));
	hipMalloc(&dev_vec, (vec2.size()+vec3.size()) * sizeof(float));
	/* hipMemcpy: �������˺��豸�˿�������,�˺������ĸ���������������֮һ:
	(1). hipMemcpyHostToHost: �������ݴ������˵�������
	(2). hipMemcpyHostToDevice: �������ݴ������˵��豸��
	(3). hipMemcpyDeviceToHost: �������ݴ��豸�˵�������
	(4). hipMemcpyDeviceToDevice: �������ݴ��豸�˵��豸��
	(5). hipMemcpyDefault: ��ָ��ֵ�Զ��ƶϿ������ݷ���,��Ҫ֧��
	ͳһ����Ѱַ(CUDA6.0�����ϰ汾)
	hipMemcpy��������������ͬ���� */
	hipMemcpy(dev_dst, dst, length * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_vec, vec2.data(), vec2.size() * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_vec + vec2.size(), vec3.data(), vec3.size() * sizeof(float), hipMemcpyHostToDevice);

	int layer_width = (int)vec1[0];
	int layer_height = (int)vec1[1];
	int image_width = (int)vec1[2];
	int image_height = (int)vec1[3];
	float offset = vec1[4];
	float step = vec1[5];
	int num_priors = (int)vec1[6];
	float width = vec1[7];
	int channel_size = layer_width * layer_height * num_priors * 4;

	TIME_START_GPU

	/* dim3: ����uint3���������ʸ�����ͣ��൱����3��unsigned int������ɵ�
	�ṹ�壬�ɱ�ʾһ����ά���飬�ڶ���dim3���ͱ���ʱ������û�и�ֵ��Ԫ�ض�
	�ᱻ����Ĭ��ֵ1 */
	// Note��ÿһ���߳̿�֧�ֵ�����߳�����Ϊ1024����threads.x*threads.y����С�ڵ���1024
	dim3 threads(32, 32);
	dim3 blocks((layer_width + 31) / 32, (layer_height + 31) / 32);

	/* <<< >>>: ΪCUDA����������,ָ���߳�������߳̿�ά�ȵ�,����ִ�в�
	����CUDA������������ʱϵͳ,����˵���ں˺����е��߳�����,�Լ��߳������
	��֯��;����������Щ���������Ǵ��ݸ��豸����Ĳ���,���Ǹ�������ʱ���
	�����豸����,���ݸ��豸���뱾��Ĳ����Ƿ���Բ�����д��ݵ�,�����׼�ĺ�
	������һ��;��ͬ�����������豸���̵߳���������֯��ʽ�в�ͬ��Լ��;����
	��Ϊkernel���õ�����������������㹻�Ŀռ�,�ٵ���kernel����,������
	GPU����ʱ�ᷢ������,����Խ��� ;
	ʹ������ʱAPIʱ,��Ҫ�ڵ��õ��ں˺�����������б�ֱ����<<<Dg,Db,Ns,S>>>
	����ʽ����ִ������,���У�Dg��һ��dim3�ͱ���,��������grid��ά�Ⱥ͸���
	ά���ϵĳߴ�.���ú�Dg��,grid�н���Dg.x*Dg.y*Dg.z��block;Db��
	һ��dim3�ͱ���,��������block��ά�Ⱥ͸���ά���ϵĳߴ�.���ú�Db��,ÿ��
	block�н���Db.x*Db.y*Db.z��thread;Ns��һ��size_t�ͱ���,ָ������Ϊ�˵�
	�ö�̬����Ĺ���洢����С,��Щ��̬����Ĵ洢���ɹ�����Ϊ�ⲿ����
	(extern __shared__)�������κα���ʹ��;Ns��һ����ѡ����,Ĭ��ֵΪ0;SΪ
	hipStream_t����,�����������ں˺�����������.S��һ����ѡ����,Ĭ��ֵ0. */
	// Note: �˺�����֧�ִ������Ϊvector��data()ָ�룬��ҪhipMalloc��hipMemcpy����Ϊvector���������ڴ���
	layer_prior_vbox << <blocks, threads>> >(dev_dst, layer_width, layer_height, image_width, image_height,
		offset, step, num_priors, width, dev_vec, dev_vec + vec2.size(), channel_size);

	/* hipDeviceSynchronize: kernel���������첽��, Ϊ�˶�λ���Ƿ����, һ
	����Ҫ����hipDeviceSynchronize��������ͬ��; ����һֱ��������״̬,ֱ��
	ǰ����������������Ѿ���ȫ��ִ�����,���ǰ��ִ�е�ĳ������ʧ��,����
	����һ�����󣻵��������ж����,������֮����ĳһ����Ҫͨ��ʱ,�Ǿͱ���
	����һ�㴦����ͬ�������,��hipDeviceSynchronize���첽����
	reference: https://stackoverflow.com/questions/11888772/when-to-call-cudadevicesynchronize */
	hipDeviceSynchronize();

	TIME_END_GPU

	hipMemcpy(dst, dev_dst, length * sizeof(float), hipMemcpyDeviceToHost);

	// hipFree: �ͷ��豸����hipMalloc����������ڴ�
	hipFree(dev_dst);
	hipFree(dev_vec);

	return 0;
}
