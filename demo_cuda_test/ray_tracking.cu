#include "hip/hip_runtime.h"
#include "funset.hpp"
#include <iostream>
#include <algorithm>
#include <memory>
#include <vector>
#include <hip/hip_runtime.h> // For the CUDA runtime routines (prefixed with "cuda_")
#include <>
#include "common.hpp"

// ͨ��һ�����ݽṹ�����潨ģ
struct Sphere {
	float r, b, g;
	float radius;
	float x, y, z;
	/* __device__: ���������޶���,���������εĺ������豸��ִ�У�ֻ�ܴ��豸�ϵ��ã�
	��ֻ��������__device__��������__global__�����е��ã�__device__������֧�ֵݹ飻
	__device__�����ĺ������ڲ���������̬������__device__�����Ĳ�����Ŀ�ǲ��ɱ仯��;
	���ܶ�__device__����ȡָ�� */
	__device__ float hit(float ox, float oy, float *n)
	{
		float dx = ox - x;
		float dy = oy - y;
		if (dx*dx + dy*dy < radius*radius) {
			float dz = sqrtf(radius*radius - dx*dx - dy*dy);
			*n = dz / sqrtf(radius * radius);
			return dz + z;
		}
		return -INF;
	}
};

// method2: ʹ�ó����ڴ�
/* __constant__: ���������޶�����������__device__�޶������ã����������ı�������
���ڳ����洢���ռ䣻��Ӧ�ó��������ͬ���������ڣ�����ͨ������ʱ��������˷��ʣ�
�豸�˵������߳�Ҳ�ɷ��ʡ�__constant__����Ĭ��Ϊ�Ǿ�̬�洢��__constant__������
extern�ؼ�������Ϊ�ⲿ������__constant__����ֻ�����ļ��������������������ٺ���
����������__constant__�������ܴ�device�и�ֵ��ֻ�ܴ�host��ͨ��host����ʱ������
ֵ��__constant__���ѱ����ķ�������Ϊֻ�������ȫ���ڴ��ж�ȡ������ȣ��ӳ�����
���ж�ȡ��ͬ�����ݿ��Խ�Լ�ڴ���������ڴ����ڱ����ں˺���ִ���ڼ䲻�ᷢ����
�������ݡ�
�����ڴ棺���ڱ����ں˺���ִ���ڼ䲻�ᷢ���仯�����ݡ�NVIDIAӲ���ṩ��64KB�ĳ�
���ڴ棬���ҶԳ����ڴ��ȡ�˲�ͬ�ڱ�׼ȫ���ڴ�Ĵ���ʽ����ĳЩ����У��ó���
�ڴ����滻ȫ���ڴ�����Ч�ؼ����ڴ���� ��ĳЩ����£�ʹ�ó����ڴ潫����Ӧ�ó�
������� */
__constant__ Sphere dev_spheres[20]; // �����ڴ�, = sphere_num

/* __global__: ���������޶���;���豸������;�������˵���,��������3.2�����Ͽ�����
�豸�˵���;�����ĺ����ķ���ֵ������void����;�Դ����ͺ����ĵ������첽��,����
�豸��ȫ�����������֮ǰ�ͷ�����;�Դ����ͺ����ĵ��ñ���ָ��ִ������,��������
�豸��ִ�к���ʱ��grid��block��ά��,�Լ���ص���(������<<<   >>>�����);
a kernel,��ʾ�˺���Ϊ�ں˺���(������GPU�ϵ�CUDA���м��㺯����Ϊkernel(�ں˺�
��),�ں˺�������ͨ��__global__���������޶�������); */
__global__ static void ray_tracking(unsigned char* ptr_image, Sphere* ptr_sphere, int width, int height, int sphere_num)
{
	/* gridDim: ���ñ���,���������߳������ά��,���������߳̿���˵,���
	������һ������,���������̸߳�ÿһά�Ĵ�С,��ÿ���̸߳����߳̿������.
	һ��gridΪ��ά,Ϊdim3���ͣ�
	blockDim: ���ñ���,����˵��ÿ��block��ά����ߴ�.Ϊdim3����,����
	��block������ά���ϵĳߴ���Ϣ;���������߳̿���˵,���������һ������,
	��������߳̿���ÿһά���߳�����;
	blockIdx: ���ñ���,�����а�����ֵ���ǵ�ǰִ���豸������߳̿������;��
	��˵����ǰthread���ڵ�block������grid�е�λ��,blockIdx.xȡֵ��Χ��
	[0,gridDim.x-1],blockIdx.yȡֵ��Χ��[0, gridDim.y-1].Ϊuint3����,
	������һ��block��grid�и���ά���ϵ�������Ϣ;
	threadIdx: ���ñ���,�����а�����ֵ���ǵ�ǰִ���豸������߳�����;����
	˵����ǰthread��block�е�λ��;����߳���һά�Ŀɻ�ȡthreadIdx.x,���
	�Ƕ�ά�Ļ��ɻ�ȡthreadIdx.y,�������ά�Ļ��ɻ�ȡthreadIdx.z;Ϊuint3��
	��,������һ��thread��block�и���ά�ȵ�������Ϣ */
	// map from threadIdx/BlockIdx to pixel position
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;
	float ox{ (x - width / 2.f) };
	float oy{ (y - height / 2.f) };

	float r{ 0 }, g{ 0 }, b{ 0 };
	float maxz{ -INF };

	for (int i = 0; i < sphere_num; ++i) {
		float n;
		float t = ptr_sphere[i].hit(ox, oy, &n);
		if (t > maxz) {
			float fscale = n;
			r = ptr_sphere[i].r * fscale;
			g = ptr_sphere[i].g * fscale;
			b = ptr_sphere[i].b * fscale;
			maxz = t;
		}
	}

	ptr_image[offset * 4 + 0] = static_cast<unsigned char>(r * 255);
	ptr_image[offset * 4 + 1] = static_cast<unsigned char>(g * 255);
	ptr_image[offset * 4 + 2] = static_cast<unsigned char>(b * 255);
	ptr_image[offset * 4 + 3] = 255;
}

__global__ static void ray_tracking(unsigned char* ptr_image, int width, int height, int sphere_num)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;
	float ox{ (x - width / 2.f) };
	float oy{ (y - height / 2.f) };

	float r{ 0 }, g{ 0 }, b{ 0 };
	float maxz{ -INF };

	for (int i = 0; i < sphere_num; ++i) {
		float n;
		float t = dev_spheres[i].hit(ox, oy, &n);
		if (t > maxz) {
			float fscale = n;
			r = dev_spheres[i].r * fscale;
			g = dev_spheres[i].g * fscale;
			b = dev_spheres[i].b * fscale;
			maxz = t;
		}
	}

	ptr_image[offset * 4 + 0] = static_cast<unsigned char>(r * 255);
	ptr_image[offset * 4 + 1] = static_cast<unsigned char>(g * 255);
	ptr_image[offset * 4 + 2] = static_cast<unsigned char>(b * 255);
	ptr_image[offset * 4 + 3] = 255;
}

int ray_tracking_gpu(const float* a, const float* b, const float* c, int sphere_num, unsigned char* ptr, int width, int height, float* elapsed_time)
{
	/* hipEvent_t: CUDA event types,�ṹ������, CUDA�¼�,���ڲ���GPU��ĳ
	�������ϻ��ѵ�ʱ��,CUDA�е��¼���������һ��GPUʱ���,����CUDA�¼�����
	GPU��ʵ�ֵ�,������ǲ����ڶ�ͬʱ�����豸�������������Ļ�ϴ����ʱ */
	hipEvent_t start, stop;
	// hipEventCreate: ����һ���¼�����,�첽����
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// hipEventRecord: ��¼һ���¼�,�첽����,start��¼��ʼʱ��
	hipEventRecord(start, 0);

	const size_t length{ width * height * 4 * sizeof(unsigned char) };
	unsigned char* dev_image{ nullptr };

	std::unique_ptr<Sphere[]> spheres(new Sphere[sphere_num]);
	for (int i = 0, t = 0; i < sphere_num; ++i, t += 3) {
		spheres[i].r = a[t];
		spheres[i].g = a[t + 1];
		spheres[i].b = a[t + 2];
		spheres[i].x = b[t];
		spheres[i].y = b[t + 1];
		spheres[i].z = b[t + 2];
		spheres[i].radius = c[i];
	}

	// hipMalloc: ���豸�˷����ڴ�
	hipMalloc(&dev_image, length);

	// method1: û��ʹ�ó����ڴ�
	//Sphere* dev_spheres{ nullptr };
	//hipMalloc(&dev_spheres, sizeof(Sphere) * sphere_num);
	/* hipMemcpy: �������˺��豸�˿�������,�˺������ĸ���������������֮һ:
	(1). hipMemcpyHostToHost: �������ݴ������˵�������
	(2). hipMemcpyHostToDevice: �������ݴ������˵��豸��
	(3). hipMemcpyDeviceToHost: �������ݴ��豸�˵�������
	(4). hipMemcpyDeviceToDevice: �������ݴ��豸�˵��豸��
	(5). hipMemcpyDefault: ��ָ��ֵ�Զ��ƶϿ������ݷ���,��Ҫ֧��
	ͳһ����Ѱַ(CUDA6.0�����ϰ汾)
	hipMemcpy��������������ͬ���� */
	//hipMemcpy(dev_spheres, spheres.get(), sizeof(Sphere) * sphere_num, hipMemcpyHostToDevice);

	// method2: ʹ�ó����ڴ�
	/* hipMemcpyToSymbol: hipMemcpyToSymbol��hipMemcpy����Ϊ
	hipMemcpyHostToDeviceʱ��Ψһ��������hipMemcpyToSymbol�Ḵ�Ƶ�������
	�棬��hipMemcpy�Ḵ�Ƶ�ȫ���ڴ� */
	hipMemcpyToSymbol(HIP_SYMBOL(dev_spheres), spheres.get(), sizeof(Sphere)* sphere_num);

	const int threads_block{ 16 };
	/* dim3: ����uint3���������ʸ�����ͣ��൱����3��unsigned int������ɵ�
	�ṹ�壬�ɱ�ʾһ����ά���飬�ڶ���dim3���ͱ���ʱ������û�и�ֵ��Ԫ�ض�
	�ᱻ����Ĭ��ֵ1 */
	dim3 blocks(width / threads_block, height / threads_block);
	dim3 threads(threads_block, threads_block);

	/* <<< >>>: ΪCUDA����������,ָ���߳�������߳̿�ά�ȵ�,����ִ�в�
	����CUDA������������ʱϵͳ,����˵���ں˺����е��߳�����,�Լ��߳������
	��֯��;����������Щ���������Ǵ��ݸ��豸����Ĳ���,���Ǹ�������ʱ���
	�����豸����,���ݸ��豸���뱾��Ĳ����Ƿ���Բ�����д��ݵ�,�����׼�ĺ�
	������һ��;��ͬ�����������豸���̵߳���������֯��ʽ�в�ͬ��Լ��;����
	��Ϊkernel���õ�����������������㹻�Ŀռ�,�ٵ���kernel����,������
	GPU����ʱ�ᷢ������,����Խ���;
	ʹ������ʱAPIʱ,��Ҫ�ڵ��õ��ں˺�����������б�ֱ����<<<Dg,Db,Ns,S>>>
	����ʽ����ִ������,���У�Dg��һ��dim3�ͱ���,��������grid��ά�Ⱥ͸���
	ά���ϵĳߴ�.���ú�Dg��,grid�н���Dg.x*Dg.y*Dg.z��block;Db��
	һ��dim3�ͱ���,��������block��ά�Ⱥ͸���ά���ϵĳߴ�.���ú�Db��,ÿ��
	block�н���Db.x*Db.y*Db.z��thread;Ns��һ��size_t�ͱ���,ָ������Ϊ�˵�
	�ö�̬����Ĺ���洢����С,��Щ��̬����Ĵ洢���ɹ�����Ϊ�ⲿ����
	(extern __shared__)�������κα���ʹ��;Ns��һ����ѡ����,Ĭ��ֵΪ0;SΪ
	hipStream_t����,�����������ں˺�����������.S��һ����ѡ����,Ĭ��ֵ0. */
	//ray_tracking << <blocks, threads >> >(dev_image, dev_spheres, width, height, sphere_num); // method1, ��ʹ�ó����ڴ�
	ray_tracking << <blocks, threads >> >(dev_image, width, height, sphere_num); // method2, ʹ�ó����ڴ�

	hipMemcpy(ptr, dev_image, length, hipMemcpyDeviceToHost);

	// hipFree: �ͷ��豸����hipMalloc����������ڴ�
	hipFree(dev_image);
	//hipFree(dev_spheres); // ʹ��method1ʱ��Ҫ�ͷ�, ���ʹ�ó����ڴ漴method2����Ҫ�ͷ�

	// hipEventRecord: ��¼һ���¼�,�첽����,stop��¼����ʱ��
	hipEventRecord(stop, 0);
	// hipEventSynchronize: �¼�ͬ��,�ȴ�һ���¼����,�첽����
	hipEventSynchronize(stop);
	// cudaEventElapseTime: ���������¼�֮�侭����ʱ��,��λΪ����,�첽����
	hipEventElapsedTime(elapsed_time, start, stop);
	// hipEventDestroy: �����¼�����,�첽����
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}
