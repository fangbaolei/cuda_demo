#include "hip/hip_runtime.h"
#include "funset.hpp"
#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>
#include <>
#include "common.hpp"

namespace {
/* __global__: ���������޶���;���豸������;�������˵���,��������3.2�����Ͽ�����
�豸�˵���;�����ĺ����ķ���ֵ������void����;�Դ����ͺ����ĵ������첽��,����
�豸��ȫ�����������֮ǰ�ͷ�����;�Դ����ͺ����ĵ��ñ���ָ��ִ������,��������
�豸��ִ�к���ʱ��grid��block��ά��,�Լ���ص���(������<<<   >>>�����);
a kernel,��ʾ�˺���Ϊ�ں˺���(������GPU�ϵ�CUDA���м��㺯����Ϊkernel(�ں˺�
��),�ں˺�������ͨ��__global__���������޶�������);*/
__global__ void histogram(const unsigned char* src, int length, int* dst)
{
	/* __shared__: ���������޶�����ʹ��__shared__�޶�����������__device__��
	�������ã���ʱ�����ı���λ��block�еĹ���洢���ռ��У���block������ͬ
	���������ڣ�����ͨ��block�ڵ������̷߳��ʣ�__shared__��__constant__����
	Ĭ��Ϊ�Ǿ�̬�洢����__shared__ǰ���Լ�extern�ؼ��֣�����ʾ���Ǳ�����С
	��ִ�в���ȷ����__shared__����������ʱ���ܳ�ʼ�������Խ�CUDA C�Ĺؼ���
	__shared__��ӵ����������У��⽫ʹ�������פ���ڹ����ڴ��У�CUDA C����
	���Թ����ڴ��еı�������ͨ�������ֱ��ȡ��ͬ�Ĵ���ʽ */
	// clear out the accumulation buffer called temp since we are launched with
	// 256 threads, it is easy to clear that memory with one write per thread
	__shared__ int temp[256]; // �����ڴ滺����
	temp[threadIdx.x] = 0;
	/* __syncthreads: ���߳̿��е��߳̽���ͬ����CUDA�ܹ���ȷ���������߳̿�
	�е�ÿ���̶߳�ִ����__syncthreads()������û���κ��߳���ִ��
	__syncthreads()֮���ָ��;��ͬһ��block�е��߳�ͨ������洢��(shared
	memory)�������ݣ���ͨ��դ��ͬ��(������kernel��������Ҫͬ����λ�õ���
	__syncthreads()����)��֤�̼߳��ܹ���ȷ�ع������ݣ�ʹ��clock()������ʱ��
	���ں˺�����Ҫ������һ�δ���Ŀ�ʼ�ͽ�����λ�÷ֱ����һ��clock()������
	���������¼���������ڵ���__syncthreads()������һ��block�е�����
	thread��Ҫ��ʱ������ͬ�ģ����ֻ��Ҫ��¼ÿ��blockִ����Ҫ��ʱ������ˣ�
	������Ҫ��¼ÿ��thread��ʱ�� */
	__syncthreads();

	/* gridDim: ���ñ���,���������߳������ά��,���������߳̿���˵,���
	������һ������,���������̸߳�ÿһά�Ĵ�С,��ÿ���̸߳����߳̿������.
	Ϊdim3���ͣ�
	blockDim: ���ñ���,����˵��ÿ��block��ά����ߴ�.Ϊdim3����,����
	��block������ά���ϵĳߴ���Ϣ;���������߳̿���˵,���������һ������,
	��������߳̿���ÿһά���߳�����;
	blockIdx: ���ñ���,�����а�����ֵ���ǵ�ǰִ���豸������߳̿������;��
	��˵����ǰthread���ڵ�block������grid�е�λ��,blockIdx.xȡֵ��Χ��
	[0,gridDim.x-1],blockIdx.yȡֵ��Χ��[0, gridDim.y-1].Ϊuint3����,
	������һ��block��grid�и���ά���ϵ�������Ϣ;
	threadIdx: ���ñ���,�����а�����ֵ���ǵ�ǰִ���豸������߳�����;����
	˵����ǰthread��block�е�λ��;����߳���һά�Ŀɻ�ȡthreadIdx.x,���
	�Ƕ�ά�Ļ��ɻ�ȡthreadIdx.y,�������ά�Ļ��ɻ�ȡthreadIdx.z;Ϊuint3��
	��,������һ��thread��block�и���ά�ȵ�������Ϣ */
	// calculate the starting index and the offset to the next block that each thread will be processing
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	while (i < length) {
		/* atomicAdd: ԭ�Ӳ���,�ײ�Ӳ����ȷ����ִ����Щԭ�Ӳ���ʱ����
		���κ��̶߳������ȡ��д���ַaddr�ϵ�ֵ��ԭ�Ӻ���(atomic
		function)��λ��ȫ�ֻ���洢����һ��32λ��64λ��ִ��
		read-modify-write��ԭ�Ӳ�����Ҳ����˵��������߳�ͬʱ����ȫ�ֻ�
		����洢����ͬһλ��ʱ����֤ÿ���߳��ܹ�ʵ�ֶԹ����д���ݵĻ�
		���������һ���������֮ǰ�������κ��̶߳��޷����ʴ˵�ַ��֮��
		�Խ���һ���̳�Ϊԭ�Ӳ���������Ϊÿ���̵߳Ĳ���������Ӱ�쵽����
		�̡߳����仰˵��ԭ�Ӳ����ܹ���֤��һ����ַ�ĵ�ǰ�������֮ǰ��
		�����̶߳����ܷ��������ַ��
		atomicAdd(addr,y)��������һ��ԭ�ӵĲ������У�����������а�����
		ȡ��ַaddr����ֵ����y���ӵ����ֵ���Լ����������ص�ַaddr�� */
		atomicAdd(&temp[src[i]], 1);
		i += stride;
	}

	// sync the data from the above writes to shared memory then add the shared memory values to the values from
	// the other thread blocks using global memory atomic adds same as before, since we have 256 threads,
	// updating the global histogram is just one write per thread!
	__syncthreads();
	// ��ÿ���߳̿��ֱ��ͼ�ϲ�Ϊ�������յ�ֱ��ͼ
	atomicAdd(&(dst[threadIdx.x]), temp[threadIdx.x]);
}

__global__ void equalization(const unsigned char* src, int length, unsigned char* dst)
{

}

} // namespace

int histogram_equalization_gpu(const unsigned char* src, int width, int height, unsigned char* dst, float* elapsed_time)
{
	const int hist_sz{ 256 }, length{ width * height }, byte_sz{ (int)sizeof(unsigned char) * length};
	unsigned char *dev_src{ nullptr }, *dev_dst{ nullptr };
	int* dev_hist{ nullptr };

	// hipMalloc: ���豸�˷����ڴ�
	hipMalloc(&dev_src, byte_sz);
	hipMalloc(&dev_dst, byte_sz);
	hipMalloc(&dev_hist, hist_sz * sizeof(int));
	/* hipMemcpy: �������˺��豸�˿�������,�˺������ĸ���������������֮һ:
	(1). hipMemcpyHostToHost: �������ݴ������˵�������
	(2). hipMemcpyHostToDevice: �������ݴ������˵��豸��
	(3). hipMemcpyDeviceToHost: �������ݴ��豸�˵�������
	(4). hipMemcpyDeviceToDevice: �������ݴ��豸�˵��豸��
	(5). hipMemcpyDefault: ��ָ��ֵ�Զ��ƶϿ������ݷ���,��Ҫ֧��
	ͳһ����Ѱַ(CUDA6.0�����ϰ汾)
	hipMemcpy��������������ͬ���� */
	hipMemcpy(dev_src, src, byte_sz, hipMemcpyHostToDevice);

	/* hipMemset: �洢����ʼ������,��GPU�ڴ���ִ�С���ָ����ֵ��ʼ��������
	�豸�ڴ� */
	hipMemset(dev_hist, 0, hist_sz * sizeof(int));

	// hipDeviceProp_t: cuda�豸���Խṹ��
	// kernel launch - 2x the number of mps gave best timing
	hipDeviceProp_t prop;
	// hipGetDeviceProperties: ��ȡGPU�豸�����Ϣ
	hipGetDeviceProperties(&prop, 0);
	// hipDeviceProp_t::multiProcessorCount: �豸�϶ദ����������
	int blocks = prop.multiProcessorCount;

	TIME_START_GPU

	/* <<< >>>: ΪCUDA����������,ָ���߳�������߳̿�ά�ȵ�,����ִ�в�
	����CUDA������������ʱϵͳ,����˵���ں˺����е��߳�����,�Լ��߳������
	��֯��;����������Щ���������Ǵ��ݸ��豸����Ĳ���,���Ǹ�������ʱ���
	�����豸����,���ݸ��豸���뱾��Ĳ����Ƿ���Բ�����д��ݵ�,�����׼�ĺ�
	������һ��;��ͬ�����������豸���̵߳���������֯��ʽ�в�ͬ��Լ��;����
	��Ϊkernel���õ�����������������㹻�Ŀռ�,�ٵ���kernel����,������
	GPU����ʱ�ᷢ������,����Խ���;
	ʹ������ʱAPIʱ,��Ҫ�ڵ��õ��ں˺�����������б�ֱ����<<<Dg,Db,Ns,S>>>
	����ʽ����ִ������,���У�Dg��һ��dim3�ͱ���,��������grid��ά�Ⱥ͸���
	ά���ϵĳߴ�.���ú�Dg��,grid�н���Dg.x*Dg.y*Dg.z��block;Db��
	һ��dim3�ͱ���,��������block��ά�Ⱥ͸���ά���ϵĳߴ�.���ú�Db��,ÿ��
	block�н���Db.x*Db.y*Db.z��thread;Ns��һ��unsigned int�ͱ���,ָ������Ϊ�˵�
	�ö�̬����Ĺ���洢����С,��Щ��̬����Ĵ洢���ɹ�����Ϊ�ⲿ����
	(extern __shared__)�������κα���ʹ��;Ns��һ����ѡ����,Ĭ��ֵΪ0;SΪ
	hipStream_t����,�����������ں˺�����������.S��һ����ѡ����,Ĭ��ֵ0. */
	// ���߳̿������ΪGPU�д�����������2��ʱ�����ﵽ��������
	// Note: �˺�����֧�ִ������Ϊvector��data()ָ�룬��ҪhipMalloc��hipMemcpy����Ϊvector���������ڴ���
	histogram << <blocks * 2, 256 >> >(dev_src, length, dev_hist);

	TIME_END_GPU

	hipMemcpy(dst, dev_dst, byte_sz, hipMemcpyDeviceToHost);

	// hipFree: �ͷ��豸����hipMalloc����������ڴ�
	hipFree(dev_src);
	hipFree(dev_hist);
	hipFree(dev_dst);

	return 0;
}

