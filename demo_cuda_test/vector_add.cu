#include "hip/hip_runtime.h"
#include "funset.hpp"
#include <iostream>
#include <hip/hip_runtime.h> // For the CUDA runtime routines (prefixed with "cuda_")
#include <>
#include "common.hpp"

// reference: C:\ProgramData\NVIDIA Corporation\CUDA Samples\v8.0\0_Simple\vectorAdd
/* __global__: ���������޶���;���豸������;�������˵���,��������3.2�����Ͽ�����
�豸�˵���;�����ĺ����ķ���ֵ������void����;�Դ����ͺ����ĵ������첽��,����
�豸��ȫ�����������֮ǰ�ͷ�����;�Դ����ͺ����ĵ��ñ���ָ��ִ������,��������
�豸��ִ�к���ʱ��grid��block��ά��,�Լ���ص���(������<<<   >>>�����);
a kernel,��ʾ�˺���Ϊ�ں˺���(������GPU�ϵ�CUDA���м��㺯����Ϊkernel(�ں˺�
��),�ں˺�������ͨ��__global__���������޶�������);*/
__global__ static void vector_add(const float *A, const float *B, float *C, int numElements)
{
  /* gridDim: ���ñ���,���������߳������ά��,���������߳̿���˵,���
  ������һ������,���������̸߳�ÿһά�Ĵ�С,��ÿ���̸߳����߳̿������.
  һ��gridΪ��ά,Ϊdim3���ͣ�
  blockDim: ���ñ���,����˵��ÿ��block��ά����ߴ�.Ϊdim3����,����
  ��block������ά���ϵĳߴ���Ϣ;���������߳̿���˵,���������һ������,
  ��������߳̿���ÿһά���߳�����;
  blockIdx: ���ñ���,�����а�����ֵ���ǵ�ǰִ���豸������߳̿������;��
  ��˵����ǰthread���ڵ�block������grid�е�λ��,blockIdx.xȡֵ��Χ��
  [0,gridDim.x-1],blockIdx.yȡֵ��Χ��[0, gridDim.y-1].Ϊuint3����,
  ������һ��block��grid�и���ά���ϵ�������Ϣ;
  threadIdx: ���ñ���,�����а�����ֵ���ǵ�ǰִ���豸������߳�����;����
  ˵����ǰthread��block�е�λ��;����߳���һά�Ŀɻ�ȡthreadIdx.x,���
  �Ƕ�ά�Ļ��ɻ�ȡthreadIdx.y,�������ά�Ļ��ɻ�ȡthreadIdx.z;Ϊuint3��
  ��,������һ��thread��block�и���ά�ȵ�������Ϣ */
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < numElements) {
    C[i] = A[i] + B[i];
  }
}

int vector_add_gpu(const float* A, const float* B, float* C, int numElements, float* elapsed_time)
{
  /* Error code to check return values for CUDA calls
  hipError_t: CUDA Error types, ö������,CUDA������,�ɹ�����
  hipSuccess(0),���򷵻�����(>0) */
  hipError_t err{ hipSuccess };

  /* hipEvent_t: CUDA event types���ṹ������, CUDA�¼������ڲ���GPU��ĳ
  �������ϻ��ѵ�ʱ�䣬CUDA�е��¼���������һ��GPUʱ���������CUDA�¼�����
  GPU��ʵ�ֵģ�������ǲ����ڶ�ͬʱ�����豸�������������Ļ�ϴ����ʱ*/
  hipEvent_t start, stop;
  // hipEventCreate: ����һ���¼������첽����
  hipEventCreate(&start);
  hipEventCreate(&stop);
  // hipEventRecord: ��¼һ���¼����첽����,start��¼��ʼʱ��
  hipEventRecord(start, 0);

  size_t length{ numElements * sizeof(float) };
  float *d_A{ nullptr }, *d_B{ nullptr }, *d_C{ nullptr };

  // hipMalloc: ���豸�˷����ڴ�
  err = hipMalloc(&d_A, length);
  if (err != hipSuccess) {
    // hipGetErrorString: ���ش�����������ַ���
    fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n",
      hipGetErrorString(err));
    return -1;
  }
  err = hipMalloc(&d_B, length);
  if (err != hipSuccess) PRINT_ERROR_INFO(hipMalloc);
  err = hipMalloc(&d_C, length);
  if (err != hipSuccess) PRINT_ERROR_INFO(hipMalloc);

  /* hipMemcpy: �������˺��豸�˿�������,�˺������ĸ���������������֮һ:
  (1). hipMemcpyHostToHost: �������ݴ������˵�������
  (2). hipMemcpyHostToDevice: �������ݴ������˵��豸��
  (3). hipMemcpyDeviceToHost: �������ݴ��豸�˵�������
  (4). hipMemcpyDeviceToDevice: �������ݴ��豸�˵��豸��
  (5). hipMemcpyDefault: ��ָ��ֵ�Զ��ƶϿ������ݷ���,��Ҫ֧��
  ͳһ����Ѱַ(CUDA6.0�����ϰ汾)
  hipMemcpy��������������ͬ���� */
  err = hipMemcpy(d_A, A, length, hipMemcpyHostToDevice);
  if (err != hipSuccess) PRINT_ERROR_INFO(hipMemcpy);
  err = hipMemcpy(d_B, B, length, hipMemcpyHostToDevice);
  if (err != hipSuccess) PRINT_ERROR_INFO(hipMemcpy);

  // Launch the Vector Add CUDA kernel
  const int threadsPerBlock{ 256 };
  const int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
  fprintf(stderr, "CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
  /* <<< >>>: ΪCUDA����������,ָ���߳�������߳̿�ά�ȵ�,����ִ�в�
  ����CUDA������������ʱϵͳ,����˵���ں˺����е��߳�����,�Լ��߳������
  ��֯��;����������Щ���������Ǵ��ݸ��豸����Ĳ���,���Ǹ�������ʱ���
  �����豸����,���ݸ��豸���뱾��Ĳ����Ƿ���Բ�����д��ݵ�,�����׼�ĺ�
  ������һ��;��ͬ�����������豸���̵߳���������֯��ʽ�в�ͬ��Լ��;����
  ��Ϊkernel���õ�����������������㹻�Ŀռ�,�ٵ���kernel����,������
  GPU����ʱ�ᷢ������,����Խ���;
  ʹ������ʱAPIʱ,��Ҫ�ڵ��õ��ں˺�����������б�ֱ����<<<Dg,Db,Ns,S>>>
  ����ʽ����ִ������,���У�Dg��һ��dim3�ͱ���,��������grid��ά�Ⱥ͸���
  ά���ϵĳߴ�.���ú�Dg��,grid�н���Dg.x*Dg.y*Dg.z��block;Db��
  һ��dim3�ͱ���,��������block��ά�Ⱥ͸���ά���ϵĳߴ�.���ú�Db��,ÿ��
  block�н���Db.x*Db.y*Db.z��thread;Ns��һ��size_t�ͱ���,ָ������Ϊ�˵�
  �ö�̬����Ĺ���洢����С,��Щ��̬����Ĵ洢���ɹ�����Ϊ�ⲿ����
  (extern __shared__)�������κα���ʹ��;Ns��һ����ѡ����,Ĭ��ֵΪ0;SΪ
  hipStream_t����,�����������ں˺�����������.S��һ����ѡ����,Ĭ��ֵ0. */
  vector_add << <blocksPerGrid, threadsPerBlock >> >(d_A, d_B, d_C, numElements);
  /* hipGetLastError: ��ͬһ�������߳���,��������ʱ�����в��������һ��
  ���󲢽�������ΪhipSuccess;�˺���Ҳ���ܷ�����ǰ�첽�����Ĵ�����;����
  ��������ڶ�hipGetLastError�ĵ���֮�䷢��ʱ,�����һ������ᱻ����;
  kernel���������첽��,Ϊ�˶�λ���Ƿ����,һ����Ҫ����
  hipDeviceSynchronize��������ͬ��,Ȼ���ٵ���hipGetLastError����;*/
  err = hipGetLastError();
  if (err != hipSuccess) PRINT_ERROR_INFO(hipGetLastError);
  // Copy the device result vector in device memory to the host result vector in host memory.
  err = hipMemcpy(C, d_C, length, hipMemcpyDeviceToHost);
  if (err != hipSuccess) PRINT_ERROR_INFO(hipMemcpy);

  // hipFree: �ͷ��豸����hipMalloc����������ڴ�
  err = hipFree(d_A);
  if (err != hipSuccess) PRINT_ERROR_INFO(hipFree);
  err = hipFree(d_B);
  if (err != hipSuccess) PRINT_ERROR_INFO(hipFree);
  err = hipFree(d_C);
  if (err != hipSuccess) PRINT_ERROR_INFO(hipFree);

  // hipEventRecord: ��¼һ���¼����첽����,stop��¼����ʱ��
  hipEventRecord(stop, 0);
  // hipEventSynchronize: �¼�ͬ�����ȴ�һ���¼���ɣ��첽����
  hipEventSynchronize(stop);
  // cudaEventElapseTime: ���������¼�֮�侭����ʱ�䣬��λΪ���룬�첽����
  hipEventElapsedTime(elapsed_time, start, stop);
  // hipEventDestroy: �����¼������첽����
  hipEventDestroy(start);
  hipEventDestroy(stop);

  return err;
}
