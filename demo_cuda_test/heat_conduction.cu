#include "hip/hip_runtime.h"
#include "funset.hpp"
#include <iostream>
#include <algorithm>
#include <memory>
#include <vector>
#include <hip/hip_runtime.h> // For the CUDA runtime routines (prefixed with "cuda_")
#include <>
#include "common.hpp"

/* __global__: ���������޶���;���豸������;�������˵���,��������3.2�����Ͽ�����
�豸�˵���;�����ĺ����ķ���ֵ������void����;�Դ����ͺ����ĵ������첽��,����
�豸��ȫ�����������֮ǰ�ͷ�����;�Դ����ͺ����ĵ��ñ���ָ��ִ������,��������
�豸��ִ�к���ʱ��grid��block��ά��,�Լ���ص���(������<<<   >>>�����);
a kernel,��ʾ�˺���Ϊ�ں˺���(������GPU�ϵ�CUDA���м��㺯����Ϊkernel(�ں˺�
��),�ں˺�������ͨ��__global__���������޶�������); */
__global__ static void copy_const_kernel(float* iptr, const float* cptr)
{
	/* gridDim: ���ñ���,���������߳������ά��,���������߳̿���˵,���
	������һ������,���������̸߳�ÿһά�Ĵ�С,��ÿ���̸߳����߳̿������.
	һ��gridΪ��ά,Ϊdim3���ͣ�
	blockDim: ���ñ���,����˵��ÿ��block��ά����ߴ�.Ϊdim3����,����
	��block������ά���ϵĳߴ���Ϣ;���������߳̿���˵,���������һ������,
	��������߳̿���ÿһά���߳�����;
	blockIdx: ���ñ���,�����а�����ֵ���ǵ�ǰִ���豸������߳̿������;��
	��˵����ǰthread���ڵ�block������grid�е�λ��,blockIdx.xȡֵ��Χ��
	[0,gridDim.x-1],blockIdx.yȡֵ��Χ��[0, gridDim.y-1].Ϊuint3����,
	������һ��block��grid�и���ά���ϵ�������Ϣ;
	threadIdx: ���ñ���,�����а�����ֵ���ǵ�ǰִ���豸������߳�����;����
	˵����ǰthread��block�е�λ��;����߳���һά�Ŀɻ�ȡthreadIdx.x,���
	�Ƕ�ά�Ļ��ɻ�ȡthreadIdx.y,�������ά�Ļ��ɻ�ȡthreadIdx.z;Ϊuint3��
	��,������һ��thread��block�и���ά�ȵ�������Ϣ */
	// map from threadIdx/BlockIdx to pixel position
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	if (cptr[offset] != 0) iptr[offset] = cptr[offset];
}

__global__ static void blend_kernel(float* outSrc, const float* inSrc, int width, int height, float speed)
{
	// map from threadIdx/BlockIdx to pixel position
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	int left = offset - 1;
	int right = offset + 1;
	if (x == 0) ++left;
	if (x == width - 1) --right;

	int top = offset - height;
	int bottom = offset + height;
	if (y == 0) top += height;
	if (y == height - 1) bottom -= height;

	outSrc[offset] = inSrc[offset] + speed * (inSrc[top] + inSrc[bottom] + inSrc[left] + inSrc[right] - inSrc[offset] * 4);
}

/* __device__: ���������޶���,���������εĺ������豸��ִ�У�ֻ�ܴ��豸�ϵ��ã�
��ֻ��������__device__��������__global__�����е��ã�__device__������֧�ֵݹ飻
__device__�����ĺ������ڲ���������̬������__device__�����Ĳ�����Ŀ�ǲ��ɱ仯��;
���ܶ�__device__����ȡָ�� */
__device__ static unsigned char value(float n1, float n2, int hue)
{
	if (hue > 360) hue -= 360;
	else if (hue < 0) hue += 360;

	if (hue < 60)
		return (unsigned char)(255 * (n1 + (n2 - n1)*hue / 60));
	if (hue < 180)
		return (unsigned char)(255 * n2);
	if (hue < 240)
		return (unsigned char)(255 * (n1 + (n2 - n1)*(240 - hue) / 60));
	return (unsigned char)(255 * n1);
}

__global__ static void float_to_color(unsigned char *optr, const float *outSrc)
{
	// map from threadIdx/BlockIdx to pixel position
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	float l = outSrc[offset];
	float s = 1;
	int h = (180 + (int)(360.0f * outSrc[offset])) % 360;
	float m1, m2;

	if (l <= 0.5f) m2 = l * (1 + s);
	else m2 = l + s - l * s;
	m1 = 2 * l - m2;

	optr[offset * 4 + 0] = value(m1, m2, h + 120);
	optr[offset * 4 + 1] = value(m1, m2, h);
	optr[offset * 4 + 2] = value(m1, m2, h - 120);
	optr[offset * 4 + 3] = 255;
}

static int heat_conduction_gpu_1(unsigned char* ptr, int width, int height, const float* src, float speed, float* elapsed_time)
{
	/* hipEvent_t: CUDA event types,�ṹ������, CUDA�¼�,���ڲ���GPU��ĳ
	�������ϻ��ѵ�ʱ��,CUDA�е��¼���������һ��GPUʱ���,����CUDA�¼�����
	GPU��ʵ�ֵ�,������ǲ����ڶ�ͬʱ�����豸�������������Ļ�ϴ����ʱ */
	hipEvent_t start, stop;
	// hipEventCreate: ����һ���¼�����,�첽����
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// hipEventRecord: ��¼һ���¼�,�첽����,start��¼��ʼʱ��
	hipEventRecord(start, 0);

	float* dev_inSrc{ nullptr };
	float* dev_outSrc{ nullptr };
	float* dev_constSrc{ nullptr };
	unsigned char* dev_image{ nullptr };
	const size_t length1{ width * height * sizeof(float) };
	const size_t length2{ width * height * 4 * sizeof(unsigned char) };

	// hipMalloc: ���豸�˷����ڴ�
	hipMalloc(&dev_inSrc, length1);
	hipMalloc(&dev_outSrc, length1);
	hipMalloc(&dev_constSrc, length1);
	hipMalloc(&dev_image, length2);

	/* hipMemcpy: �������˺��豸�˿�������,�˺������ĸ���������������֮һ:
	(1). hipMemcpyHostToHost: �������ݴ������˵�������
	(2). hipMemcpyHostToDevice: �������ݴ������˵��豸��
	(3). hipMemcpyDeviceToHost: �������ݴ��豸�˵�������
	(4). hipMemcpyDeviceToDevice: �������ݴ��豸�˵��豸��
	(5). hipMemcpyDefault: ��ָ��ֵ�Զ��ƶϿ������ݷ���,��Ҫ֧��
	ͳһ����Ѱַ(CUDA6.0�����ϰ汾)
	hipMemcpy��������������ͬ���� */
	hipMemcpy(dev_constSrc, src, length1, hipMemcpyHostToDevice);

	const int threads_block{ 16 };
	/* dim3: ����uint3���������ʸ�����ͣ��൱����3��unsigned int������ɵ�
	�ṹ�壬�ɱ�ʾһ����ά���飬�ڶ���dim3���ͱ���ʱ������û�и�ֵ��Ԫ�ض�
	�ᱻ����Ĭ��ֵ1 */
	dim3 blocks(width / threads_block, height / threads_block);
	dim3 threads(threads_block, threads_block);

	for (int i = 0; i < 90; ++i) {
		copy_const_kernel << <blocks, threads >> >(dev_inSrc, dev_constSrc);
		blend_kernel << <blocks, threads >> >(dev_outSrc, dev_inSrc, width, height, speed);
		std::swap(dev_inSrc, dev_outSrc);
	}

	/* <<< >>>: ΪCUDA����������,ָ���߳�������߳̿�ά�ȵ�,����ִ�в�
	����CUDA������������ʱϵͳ,����˵���ں˺����е��߳�����,�Լ��߳������
	��֯��;����������Щ���������Ǵ��ݸ��豸����Ĳ���,���Ǹ�������ʱ���
	�����豸����,���ݸ��豸���뱾��Ĳ����Ƿ���Բ�����д��ݵ�,�����׼�ĺ�
	������һ��;��ͬ�����������豸���̵߳���������֯��ʽ�в�ͬ��Լ��;����
	��Ϊkernel���õ�����������������㹻�Ŀռ�,�ٵ���kernel����,������
	GPU����ʱ�ᷢ������,����Խ���;
	ʹ������ʱAPIʱ,��Ҫ�ڵ��õ��ں˺�����������б�ֱ����<<<Dg,Db,Ns,S>>>
	����ʽ����ִ������,���У�Dg��һ��dim3�ͱ���,��������grid��ά�Ⱥ͸���
	ά���ϵĳߴ�.���ú�Dg��,grid�н���Dg.x*Dg.y*Dg.z��block;Db��
	һ��dim3�ͱ���,��������block��ά�Ⱥ͸���ά���ϵĳߴ�.���ú�Db��,ÿ��
	block�н���Db.x*Db.y*Db.z��thread;Ns��һ��size_t�ͱ���,ָ������Ϊ�˵�
	�ö�̬����Ĺ���洢����С,��Щ��̬����Ĵ洢���ɹ�����Ϊ�ⲿ����
	(extern __shared__)�������κα���ʹ��;Ns��һ����ѡ����,Ĭ��ֵΪ0;SΪ
	hipStream_t����,�����������ں˺�����������.S��һ����ѡ����,Ĭ��ֵ0. */
	float_to_color << <blocks, threads >> >(dev_image, dev_inSrc);

	hipMemcpy(ptr, dev_image, length2, hipMemcpyDeviceToHost);

	// hipFree: �ͷ��豸����hipMalloc����������ڴ�
	hipFree(dev_inSrc);
	hipFree(dev_outSrc);
	hipFree(dev_constSrc);
	hipFree(dev_image);

	// hipEventRecord: ��¼һ���¼�,�첽����,stop��¼����ʱ��
	hipEventRecord(stop, 0);
	// hipEventSynchronize: �¼�ͬ��,�ȴ�һ���¼����,�첽����
	hipEventSynchronize(stop);
	// cudaEventElapseTime: ���������¼�֮�侭����ʱ��,��λΪ����,�첽����
	hipEventElapsedTime(elapsed_time, start, stop);
	// hipEventDestroy: �����¼�����,�첽����
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}

static int heat_conduction_gpu_2(unsigned char* ptr, int width, int height, const float* src, float speed, float* elapsed_time)
{
	return 0;
}

static int heat_conduction_gpu_3(unsigned char* ptr, int width, int height, const float* src, float speed, float* elapsed_time)
{
	return 0;
}

int heat_conduction_gpu(unsigned char* ptr, int width, int height, const float* src, float speed, float* elapsed_time)
{
	int ret{ 0 };
	ret = heat_conduction_gpu_1(ptr, width, height, src, speed, elapsed_time); // û�в��������ڴ�
	//ret = heat_conduction_gpu_2(ptr, width, height, src, speed, elapsed_time); // ����һά�����ڴ�
	//ret = heat_conduction_gpu_3(ptr, width, height, src, speed, elapsed_time); // ���ö�ά�����ڴ�

	return ret;
}
