#include "hip/hip_runtime.h"
#include "funset.hpp"
#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>
#include <>
#include "common.hpp"

/* __global__: ���������޶���;���豸������;�������˵���,��������3.2�����Ͽ�����
�豸�˵���;�����ĺ����ķ���ֵ������void����;�Դ����ͺ����ĵ������첽��,����
�豸��ȫ�����������֮ǰ�ͷ�����;�Դ����ͺ����ĵ��ñ���ָ��ִ������,��������
�豸��ִ�к���ʱ��grid��block��ά��,�Լ���ص���(������<<<   >>>�����);
a kernel,��ʾ�˺���Ϊ�ں˺���(������GPU�ϵ�CUDA���м��㺯����Ϊkernel(�ں˺�
��),�ں˺�������ͨ��__global__���������޶�������);*/
__global__ static void bgr2gray(const unsigned char* src, int B2Y, int G2Y, int R2Y, int shift, int width, int height, unsigned char* dst)
{
	/* gridDim: ���ñ���,���������߳������ά��,���������߳̿���˵,���
	������һ������,���������̸߳�ÿһά�Ĵ�С,��ÿ���̸߳����߳̿������.
	һ��gridΪ��ά,Ϊdim3���ͣ�
	blockDim: ���ñ���,����˵��ÿ��block��ά����ߴ�.Ϊdim3����,����
	��block������ά���ϵĳߴ���Ϣ;���������߳̿���˵,���������һ������,
	��������߳̿���ÿһά���߳�����;
	blockIdx: ���ñ���,�����а�����ֵ���ǵ�ǰִ���豸������߳̿������;��
	��˵����ǰthread���ڵ�block������grid�е�λ��,blockIdx.xȡֵ��Χ��
	[0,gridDim.x-1],blockIdx.yȡֵ��Χ��[0, gridDim.y-1].Ϊuint3����,
	������һ��block��grid�и���ά���ϵ�������Ϣ;
	threadIdx: ���ñ���,�����а�����ֵ���ǵ�ǰִ���豸������߳�����;����
	˵����ǰthread��block�е�λ��;����߳���һά�Ŀɻ�ȡthreadIdx.x,���
	�Ƕ�ά�Ļ��ɻ�ȡthreadIdx.y,�������ά�Ļ��ɻ�ȡthreadIdx.z;Ϊuint3��
	��,������һ��thread��block�и���ά�ȵ�������Ϣ */
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	//if (x == 0 && y == 0) {
	//	printf("%d, %d, %d, %d, %d, %d\n", width, height, B2Y, G2Y, R2Y, shift);
	//}

	if (x < width && y < height) {
		dst[y * width + x] = (unsigned char)((src[y*width * 3 + 3 * x + 0] * B2Y +
			src[y*width * 3 + 3 * x + 1] * G2Y + src[y*width * 3 + 3 * x + 2] * R2Y) >> shift);
	}
}

int bgr2gray_gpu(const unsigned char* src, int width, int height, unsigned char* dst, float* elapsed_time)
{
	const int R2Y{ 4899 }, G2Y{ 9617 }, B2Y{ 1868 }, yuv_shift{ 14 };
	unsigned char *dev_src{ nullptr }, *dev_dst{ nullptr };
	// hipMalloc: ���豸�˷����ڴ�
	hipMalloc(&dev_src, width * height * 3 * sizeof(unsigned char));
	hipMalloc(&dev_dst, width * height * sizeof(unsigned char));
	/* hipMemcpy: �������˺��豸�˿�������,�˺������ĸ���������������֮һ:
	(1). hipMemcpyHostToHost: �������ݴ������˵�������
	(2). hipMemcpyHostToDevice: �������ݴ������˵��豸��
	(3). hipMemcpyDeviceToHost: �������ݴ��豸�˵�������
	(4). hipMemcpyDeviceToDevice: �������ݴ��豸�˵��豸��
	(5). hipMemcpyDefault: ��ָ��ֵ�Զ��ƶϿ������ݷ���,��Ҫ֧��
	ͳһ����Ѱַ(CUDA6.0�����ϰ汾)
	hipMemcpy��������������ͬ���� */
	hipMemcpy(dev_src, src, width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice);
	/* hipMemset: �洢����ʼ������,��GPU�ڴ���ִ�С���ָ����ֵ��ʼ��������
	�豸�ڴ� */
	hipMemset(dev_dst, 0, width * height * sizeof(unsigned char));

	TIME_START_GPU

	/* dim3: ����uint3���������ʸ�����ͣ��൱����3��unsigned int������ɵ�
	�ṹ�壬�ɱ�ʾһ����ά���飬�ڶ���dim3���ͱ���ʱ������û�и�ֵ��Ԫ�ض�
	�ᱻ����Ĭ��ֵ1 */
	// Note��ÿһ���߳̿�֧�ֵ�����߳�����Ϊ1024����threads.x*threads.y����С�ڵ���1024
	dim3 threads(32, 32);
	dim3 blocks((width + 31) / 32, (height + 31) / 32);

	/* <<< >>>: ΪCUDA����������,ָ���߳�������߳̿�ά�ȵ�,����ִ�в�
	����CUDA������������ʱϵͳ,����˵���ں˺����е��߳�����,�Լ��߳������
	��֯��;����������Щ���������Ǵ��ݸ��豸����Ĳ���,���Ǹ�������ʱ���
	�����豸����,���ݸ��豸���뱾��Ĳ����Ƿ���Բ�����д��ݵ�,�����׼�ĺ�
	������һ��;��ͬ�����������豸���̵߳���������֯��ʽ�в�ͬ��Լ��;����
	��Ϊkernel���õ�����������������㹻�Ŀռ�,�ٵ���kernel����,������
	GPU����ʱ�ᷢ������,����Խ��� ;
	ʹ������ʱAPIʱ,��Ҫ�ڵ��õ��ں˺�����������б�ֱ����<<<Dg,Db,Ns,S>>>
	����ʽ����ִ������,���У�Dg��һ��dim3�ͱ���,��������grid��ά�Ⱥ͸���
	ά���ϵĳߴ�.���ú�Dg��,grid�н���Dg.x*Dg.y*Dg.z��block;Db��
	һ��dim3�ͱ���,��������block��ά�Ⱥ͸���ά���ϵĳߴ�.���ú�Db��,ÿ��
	block�н���Db.x*Db.y*Db.z��thread;Ns��һ��size_t�ͱ���,ָ������Ϊ�˵�
	�ö�̬����Ĺ���洢����С,��Щ��̬����Ĵ洢���ɹ�����Ϊ�ⲿ����
	(extern __shared__)�������κα���ʹ��;Ns��һ����ѡ����,Ĭ��ֵΪ0;SΪ
	hipStream_t����,�����������ں˺�����������.S��һ����ѡ����,Ĭ��ֵ0. */
	// Note: �˺�����֧�ִ������Ϊvector��data()ָ�룬��ҪhipMalloc��hipMemcpy����Ϊvector���������ڴ���
	bgr2gray << <blocks, threads >> >(dev_src, B2Y, G2Y, R2Y, yuv_shift, width, height, dev_dst);

	/* hipDeviceSynchronize: kernel���������첽��, Ϊ�˶�λ���Ƿ����, һ
	����Ҫ����hipDeviceSynchronize��������ͬ��; ����һֱ��������״̬,ֱ��
	ǰ����������������Ѿ���ȫ��ִ�����,���ǰ��ִ�е�ĳ������ʧ��,����
	����һ�����󣻵��������ж����,������֮����ĳһ����Ҫͨ��ʱ,�Ǿͱ���
	����һ�㴦����ͬ�������,��hipDeviceSynchronize���첽����
	reference: https://stackoverflow.com/questions/11888772/when-to-call-cudadevicesynchronize */
	hipDeviceSynchronize();

	TIME_END_GPU

	hipMemcpy(dst, dev_dst, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost);

	// hipFree: �ͷ��豸����hipMalloc����������ڴ�
	hipFree(dev_dst);
	hipFree(dev_src);

	return 0;
}
